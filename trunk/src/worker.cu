#include "../include/worker.cuh"

/* 
 * ====================
 * WorkResult
 * ====================
 */
WorkResult::WorkResult(WorkResult::RESULTS resultType, ErrorResult& results) : _resultType(resultType), _results(&results) {
}

WorkResult::WorkResult(WorkResult::RESULTS resultType) : _resultType(resultType), _results(NULL) {
}

WorkResult::~WorkResult() {
    if (_results != NULL) {
        delete _results; // delete NULL is ok
    }
}

ErrorResult& WorkResult::getResults() const {
    return *_results;
}

WorkResult::RESULTS WorkResult::getResultType() {
    return _resultType;
}

/* 
 * ====================
 * Worker
 * ====================
 */
Worker::Worker(ConvNet* convNet) : _convNet(convNet) {
}

/* 
 * ====================
 * TrainingWorker
 * ====================
 */
TrainingWorker::TrainingWorker(ConvNet* convNet, CPUData& data, bool test) 
    : Worker(convNet), _data(&data), _test(test) {
}

void TrainingWorker::run() {
    // Need to setData here (as opposed to the constructor) because the constructor executes in
    // the original CPU thread, which is not the one with GPU access.
    _convNet->getDataProvider().setData(*_data);
    ErrorResult& batchErr = *new ErrorResult();
    for (int i = 0; i < _convNet->getDataProvider().getNumMinibatches(); i++) {
        GPUData& mini = _convNet->getDataProvider()[i];

        _convNet->getLayerGraph().fprop(mini);
        ErrorResult& miniErr = _convNet->getLayerGraph().getError();
        batchErr += miniErr;

        if (!_test) {
            _convNet->getLayerGraph().bprop();
            _convNet->getLayerGraph().updateWeights();
        }

        delete &mini;
        delete &miniErr;
    }
    hipDeviceSynchronize();

    batchErr /= _convNet->getDataProvider().getNumCases();
    _convNet->getResultQueue().enqueue(new WorkResult(WorkResult::BATCH_DONE, batchErr));
}

/*
 * ====================
 * SyncWorker
 * ====================
 */
SyncWorker::SyncWorker(ConvNet* convNet) : Worker(convNet) {
}

void SyncWorker::run() {
    _convNet->getLayerGraph().copyToCPU();
    _convNet->getResultQueue().enqueue(new WorkResult(WorkResult::SYNC_DONE));
}

/* 
 * ====================
 * GradCheckWorker
 * ====================
 */
GradCheckWorker::GradCheckWorker(ConvNet* convNet, CPUData& data) 
    : Worker(convNet), _data(&data) {
}

void GradCheckWorker::run() {
    _convNet->getDataProvider().setData(*_data);
    _convNet->getLayerGraph().checkGradients(_convNet->getDataProvider().getMinibatch(0));
    exit(0);
}