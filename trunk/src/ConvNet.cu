#include "hip/hip_runtime.h"
/* 
 * Author: Alex Krizhevsky (akrizhevsky@gmail.com)
 * June 2011
 */
#include <vector>
#include <iostream> 
#include <string>
#include "../include/ConvNet.cuh"

using namespace std;

/* 
 * =======================
 * ConvNet
 * =======================
 */

ConvNet::ConvNet(PyListObject* layerParams, int minibatchSize, int deviceID) 
    : Thread(false),  _deviceID(deviceID), _data(NULL), _checkingGrads(false) {
    try {       
        int numDefs = PyList_GET_SIZE(layerParams);
    
        for (int i = 0; i < numDefs; i++) {
            PyObject* paramsDict = PyList_GET_ITEM(layerParams, i);
            char* layerType = PyString_AS_STRING(PyDict_GetItemString(paramsDict, "type"));

            if (string(layerType) == string("fc")) {
                _layers.push_back(dynamic_cast<Layer*>(new FCLayer(paramsDict, this)));
            } else if (string(layerType) == string("conv")) {
                _layers.push_back(dynamic_cast<Layer*>(new ConvLayer(paramsDict, this)));
            } else if (string(layerType) == string("pool")) {
                _layers.push_back(dynamic_cast<Layer*>(new PoolLayer(paramsDict, this)));
            } else if (string(layerType) == string("data")) {
                DataLayer *d = new DataLayer(paramsDict, this);
                _layers.push_back(dynamic_cast<Layer*>(d));
                _dataLayers.push_back(d);
            } else if (string(layerType) == string("softmax")) {
                _layers.push_back(dynamic_cast<Layer*>(new SoftmaxLayer(paramsDict, this)));
            } else if (strncmp(layerType, "cost.logreg", 32) == 0) {
                Cost *c = new LogregCost(paramsDict, this);
                _layers.push_back(dynamic_cast<Layer*>(c));
                _costs.push_back(c);
            } else {
                throw string("Unknown layer type ") + string(layerType);
            }
        }

        // Connect the forward links in the graph
        for (int i = 0; i < _layers.size(); i++) {
            vector<Layer*>& prev = _layers[i]->getPrev();
            for (int j = 0; j < prev.size(); j++) {
                prev[j]->addNext(_layers[i]);
            }
        }
        reset(); // For good measure
        
        this->_dp = new DataProvider(minibatchSize);
    } catch(string& s) {
        cout << "Error creating ConvNet: " << s << endl;
        exit(1);
    }
}

/*
 * This executes in a new CPU thread so it's OK to initialize CUDA stuff here. 
 */
void ConvNet::initCuda() { 
    hipSetDevice(_deviceID < 0 ? cutGetMaxGflopsDeviceId() : _deviceID);
    hipDeviceSetCacheConfig(hipFuncCachePreferShared);
    cublasInit();
    NVMatrix::initRandom(time(0));
    
    copyToGPU();
}

void* ConvNet::run() {
    initCuda();

    while (true) {
        Worker* worker = _workerQueue.dequeue();
        worker->run();
        delete worker;
    }
    return NULL;
}

Queue<Worker*>& ConvNet::getWorkerQueue() {
    return _workerQueue;
}

Queue<WorkResult*>& ConvNet::getResultQueue() {
    return _resultQueue;
}

DataProvider& ConvNet::getDataProvider() {
    return *_dp;
}

Layer& ConvNet::operator[](const int idx) {
    return *_layers[idx];
}

Layer& ConvNet::getLayer(const int idx) {
    return *_layers[idx];
}

void ConvNet::copyToCPU() {
    for (int i = 0; i < _layers.size(); i++) {
        _layers[i]->copyToCPU();
    }
}

void ConvNet::copyToGPU() {
    for (int i = 0; i < _layers.size(); i++) {
        _layers[i]->copyToGPU();
    }
}

void ConvNet::updateWeights() {
    for (int i = 0; i < _layers.size(); i++) {
        _layers[i]->updateWeights(_data->getNumCases());
    }
}

void ConvNet::reset() {
    for (int i = 0; i < _layers.size(); i++) {
        _layers[i]->reset();
    }
}

int ConvNet::getNumLayers() {
    return _layers.size();
}

void ConvNet::bprop() {
    for (int i = 0; i < _costs.size(); i++) {
        _costs[i]->bprop();
    }
    reset();
}

void ConvNet::fprop() {
    assert(_data != NULL);
    reset();
    for (int i = 0; i < _data->getSize(); i++) {
        _dataLayers[i]->fprop(_data->getData());
    }
}

void ConvNet::fprop(GPUData& data) {
    if (&data != _data) {
        delete _data;
    }
    _data = &data;
    fprop();
}

void ConvNet::fprop(int miniIdx) {
    delete _data;
    _data = &_dp->getMinibatch(miniIdx);
    fprop();
}

void ConvNet::setData(CPUData& data) {
    _dp->setData(data);
}

ErrorResult& ConvNet::getError() {
    return *new ErrorResult(_costs);
}

double ConvNet::getCostFunctionValue() {
    ErrorResult& err = getError();
    double val = err.getCost();
    delete &err;
    return val;
}

bool ConvNet::isCheckingGrads() {
    return _checkingGrads;
}

void ConvNet::checkGradients() {
    _checkingGrads = true;
    _numFailures = 0;
    _numTests = 0;
    fprop(0);
    _baseErr = getCostFunctionValue();
    bprop();
    
    for (vector<Layer*>::iterator it = _layers.begin(); it != _layers.end(); ++it) {
        (*it)->checkGradients();
    }
    
    cout << "------------------------" << endl;
    if (_numFailures > 0) {
        cout << _numFailures << "/" << _numTests << " TESTS FAILED" << endl;
    } else {
        cout << "ALL " << _numTests << " TESTS PASSED" << endl;
    }
    _checkingGrads = false;
}

bool ConvNet::checkGradientsW(const string& name, float eps, Weights& weights) {
    Matrix numGrads(weights.getNumRows(), weights.getNumCols());
    Matrix diff(numGrads);
    numGrads.apply(Matrix::ZERO);
    Matrix weightsCPU;

    weights.getW().copyToHost(weightsCPU, true);

    for(int i = 0; i < weights.getNumRows(); i++) {
        for (int j = 0; j < weights.getNumCols(); j++) {
            float v = weightsCPU(i,j);
            weightsCPU(i,j) += eps;
            weights.getW().copyFromHost(weightsCPU);
            weightsCPU(i,j) = v;
            fprop();
            double err = getCostFunctionValue();
            numGrads(i,j) = (err - _baseErr) / (_data->getNumCases() * eps);
            weights.getW().copyFromHost(weightsCPU);
        }
    }

    Matrix gradsCPU;
    weights.getGrads().scale(-1.0 / _data->getNumCases());
    weights.getGrads().copyToHost(gradsCPU, true);
    float analNorm = gradsCPU.norm();
    float numNorm = numGrads.norm();
    numGrads.subtract(gradsCPU, diff);
    float relErr = diff.norm() / analNorm;
    bool fail = relErr >= GC_REL_ERR_THRESH;
    if (fail || !GC_SUPPRESS_PASSES) {
        cout << "========================" << endl;
        printf("(%s) %s GRADIENT CHECK\n", fail ? "****FAIL****" : "PASS", name.c_str());
        cout << "========================" << endl;
        cout << "Analytic:" << endl;
        gradsCPU.print(6,4);
        cout << "Numeric:" << endl;
        numGrads.print(6,4);
        printf("Analytic norm: %e\n", analNorm);
        printf("Numeric norm:  %e\n", numNorm);
        printf("Relative error: %e\n", relErr);
    }
    _numTests++;
    _numFailures += fail;
    return fail;
}