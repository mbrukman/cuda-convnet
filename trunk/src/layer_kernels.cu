#include "hip/hip_runtime.h"
/* 
 * Author: Alex Krizhevsky (akrizhevsky@gmail.com)
 * June 2011
 */
#include "../include/layer_kernels.cuh"

/*
 * E = -log(y_t)
 * probs:           (numOut, caseStride)
 * labels:          (1, caseStride)
 * maxProbs:        (1, caseStride)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:  (1, numCases) == log(y_l[labels,:]
 */
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs,
                            const int numCases, const int caseStride, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * caseStride + tx];
        
        labelLogProbs[tx] = __logf(labelp);
        
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += probs[i * caseStride + tx] == maxp;
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}

