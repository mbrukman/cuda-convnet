#include "hip/hip_runtime.h"
/* 
 * Author: Alex Krizhevsky (akrizhevsky@gmail.com)
 * June 2011
 */
#include <assert.h>

#include "../include/layer_kernels.cuh"

/*
 * E = -log(y_t)
 * probs:           (numOut, numCases)
 * labels:          (1, numCases)
 * maxProbs:        (1, numCases)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:          (1, numCases) == log(y_l[labels,:]
 * 
 * numCases is the actual number of cases
 * caseStride is the number of cases including padding cases
 */
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs,
                            const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];
        
        labelLogProbs[tx] = __logf(labelp);
        
        /*
         * Compute the probability of guessing the correct case if you take the most-probable label.
         * 
         * This is done like this:
         * 
         * - If the most probable label is not equal to the true label, then the probability is zero.
         * - Otherwise, the probability is 1 / (number of labels whose probability is equal to the maximum).
         * 
         * This is certainly overkill -- in practice, it's just about impossible for two labels to get assigned
         * maximum probability. But it's a safety measure to prevent over-estimating your accuracy.
         * Though it could never happen in reality. Well it could. But it wouldn't. Cool?
         */
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += probs[i * numCases + tx] == maxp;
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}

/*
 * E = -log(y_t)
 * probs:           (numOut, numCases)
 * labels:          (1, numCases)
 * maxProbs:        (1, numCases)
 * labelLogProbs:   (1, numCases)   (*out)
 * correctProbs:    (1, numCases)   (*out)
 * 
 * target:          (1, numCases) == log(y_l[labels,:]
 */
void computeLogregCost(NVMatrix& labels, NVMatrix& probs, NVMatrix& labelLogProbs_out, NVMatrix& correctProbs_out) {
    int numCases = probs.getNumCols(); 
    int numOut = probs.getNumRows(); 

    assert(labels.getNumElements() == numCases);
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    assert(labels.isContiguous());
    assert(probs.isContiguous());
    
    NVMatrix& maxProbs = probs.max(0);
    
    labelLogProbs_out.resize(1, numCases);
    correctProbs_out.resize(1, numCases);
    dim3 threads(LOGREG_ERR_THREADS_X, 1);
    dim3 blocks(DIVUP(numCases, LOGREG_ERR_THREADS_X), 1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLogregCost), hipFuncCachePreferL1);
    kLogregCost<<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), maxProbs.getDevData(),
                                     labelLogProbs_out.getDevData(), correctProbs_out.getDevData(),
                                     numCases, numOut);
    cutilCheckMsg("kLogregCost: Kernel execution failed");
//    hipDeviceSynchronize();
    delete &maxProbs;
}

void computeLogregGrads(NVMatrix& labels, NVMatrix& probs, NVMatrix& target, bool add, float coeff) {
    int numCases = probs.getNumCols(); 
    int numOut = probs.getNumRows(); 
    assert(labels.getNumElements() == numCases);
    assert(probs.isContiguous());
    assert(target.isContiguous());
    assert(labels.isContiguous());
    assert(!labels.isTrans());
    assert(!probs.isTrans());
    
    dim3 threads(LOGREG_GRADS_THREADS_X, LOGREG_GRADS_THREADS_Y);
    dim3 blocks(DIVUP(numCases, LOGREG_GRADS_THREADS_X), DIVUP(numOut, LOGREG_GRADS_THREADS_Y));
    if (!add) {
        target.resize(probs);
        kLogregCostGrads<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                     numCases, numOut, coeff);
    } else {
        kLogregCostGrads<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                    numCases, numOut, coeff);
    }

    cutilCheckMsg("kLogregCostGrads: Kernel execution failed");
}