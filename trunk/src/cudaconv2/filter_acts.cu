#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <cutil_inline.h>
#include <nvmatrix.cuh>
#include <cudaconv2.cuh>

/*
 * Block size B_YxB_X. Each block applies B_Y * filtersPerThread filters to B_X * imgsPerThread images.
 * threadIdx.x determines image
 * threadIdx.y determines filter
 *
 * blockIdx.x determines image batch of B_X * imgsPerThread
 * blockIdx.y determines filter batch of module and B_Y * filtersPerThread
 *
 * images:      (numColors, imgPixels, numImages) with stride given
 * filters:     (numColors, filterPixels, numFilters)
 *
 * targets:     (numFilters, numModules, numImages)
 *
 * B_Y one of 4, 8, 16
 * B_X one of 16, 32
 * imgsPerThread one of 1, 2, 4
 * filtersPerThread one of 1, 2, 4, 8
 *
 * Number of filters per module should be divisible by B_Y * filtersPerThread
 * checkImgBounds indicates whether number of images is divisible by B_X * imgsPerThread
 *
 * The imgSize here is the size of the actual image without the padding.
 *
 */
template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int numColors, bool scale, bool checkImgBounds>
__global__ void filterActs_YxX_color(float* images, float* filters, float* targets,
                                   const int numImages, const int numFilters,
                                   const int imgSize, const int filterSize, const int paddingStart,
                                   const int moduleStride,
                                   const int numModulesX, const int imgStride,
                                   const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[B_Y*numColors][B_Y * filtersPerThread]; // pre-load B_Y pixels from B_Y*filtersPerThread filters
    __shared__ float shImages[B_Y*numColors][B_X * imgsPerThread]; // pre-load B_Y pixels from B_X*imgsPerThread images
    const int imgPixels = imgSize * imgSize;
    const int filterPixels = filterSize * filterSize;

    const int blocksPerModule = numFilters / (B_Y*filtersPerThread);
    const int moduleIdx = blockIdx.y / blocksPerModule;
    const int blockFilterIdx = blockIdx.y % blocksPerModule;

    const int tidx = threadIdx.y * B_X + threadIdx.x;

    const int imgLoadModPosY = (moduleIdx / numModulesX) * moduleStride;
    const int imgLoadModPosX = (moduleIdx % numModulesX) * moduleStride;

    const int shFilterLoadY = tidx / (B_Y * filtersPerThread);
    const int shFilterLoadX = tidx % (B_Y * filtersPerThread);
    const int myImgIdx = blockIdx.x * B_X * imgsPerThread + threadIdx.x;
    images += myImgIdx;
    filters += filtersPerThread * B_Y * blockFilterIdx
            + shFilterLoadY * numFilters + shFilterLoadX;

    targets += moduleIdx * numImages
            + (blockFilterIdx * B_Y * filtersPerThread + threadIdx.y) * numImages * numModulesX * numModulesX
            + myImgIdx;


    float prod[filtersPerThread][imgsPerThread];
    #pragma unroll
    for(int f = 0; f < filtersPerThread; f++) {
        #pragma unroll
        for(int g = 0; g < imgsPerThread; g++) {
            prod[f][g] = 0;
        }
    }

    for (int p = 0; p < filterPixels; p += B_Y) {
        /*
         * Load B_Y pixels from B_Y*filtersPerThread filters
         */
        if (shFilterLoadY < B_Y) {
            #pragma unroll
            for (int p2 = 0; p2 < B_Y; p2 += B_X/filtersPerThread) {
                if (p + p2 + shFilterLoadY < filterPixels) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = filters[(c * filterPixels + p + p2) * numFilters];
                    }
                } else {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = 0;
                    }
                }
            }
        }

        /*
         * Load B_Y pixels from B_X*imgsPerThread images
         */
        const int pixIdx = p + threadIdx.y;
        if (pixIdx < filterPixels) {
            const int x = paddingStart + imgLoadModPosX + pixIdx % filterSize;
            const int y = paddingStart + imgLoadModPosY + pixIdx / filterSize;
            if (y >= 0 && y< imgSize && x >= 0 && x < imgSize) {
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    if (!checkImgBounds || myImgIdx + i * B_X < numImages) {
                        #pragma unroll
                        for (int c = 0; c < numColors; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = images[imgStride * (c * imgPixels + y * imgSize + x) + i * B_X];
                        }
                    } else {
                        #pragma unroll
                        for (int c = 0; c < numColors; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                        }
                    }
                }
            } else { // Padding
                #pragma unroll
                for (int i = 0; i < imgsPerThread; i++) {
                    #pragma unroll
                    for (int c = 0; c < numColors; c++) {
                        shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                    }
                }
            }
        }
        __syncthreads();
        #pragma unroll
        for (int i = 0; i < B_Y*numColors; i++) {
            #pragma unroll
            for(int f = 0; f < filtersPerThread; f++) {
                #pragma unroll
                for(int g = 0; g < imgsPerThread; g++) {
                    prod[f][g] += shImages[i][g * B_X + threadIdx.x] * shFilters[i][threadIdx.y + f * B_Y];
                }
            }

        }
        __syncthreads();
    }
    
    if (scale) {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModulesX * numModulesX] = scaleTargets * targets[g * B_X + f * B_Y * numImages * numModulesX * numModulesX] + scaleOutputs * prod[f][g];
                }
            }
        }
    } else {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModulesX * numModulesX] = prod[f][g];
                }
            }
        }
    }
}

/*
 * Block size B_YxB_X. Each block applies B_Y * filtersPerThread filters to B_X * imgsPerThread images.
 * threadIdx.x determines image
 * threadIdx.y determines filter
 *
 * blockIdx.x determines image batch of B_X * imgsPerThread
 * blockIdx.y determines filter batch of B_Y * filtersPerThread
 *
 * images:      (numImgColors, imgPixels, numImages) with stride given
 * filters:     (numFilterColors, filterPixels, numFilters)
 *
 * targets:     (numFilters, numModules, numImages)
 *
 * B_Y one of 4, 8, 16
 * B_X one of 16, 32
 * imgsPerThread one of 1, 2, 4
 * filtersPerThread one of 1, 2, 4, 8
 * colorCache: how many colors to put into shmem
 *
 * numFilters should be divisible by B_Y * filtersPerThread
 * numImages be divisible by B_X * imgsPerThread
 * numFilterColors should be divisible by colorCache.
 * numImgColors must be even.
 * numFilters must be divisible by numGroups.
 *
 * The imgSize here is the size of the actual image without the padding.
 *
 */
template <int B_Y, int B_X, int imgsPerThread, int filtersPerThread, int colorCache, bool scale, bool checkImgBounds>
__global__ void filterActs_YxX_sparse(float* images, float* filters, float* targets,
                                       const int numImages, const int numFilters,
                                       const int imgSize, const int filterSize, const int paddingStart,
                                       const int moduleStride,
                                       const int numModulesX, const int imgStride, const int numImgColors,
                                       const int numGroups, 
                                       const float scaleTargets, const float scaleOutputs) {
    __shared__ float shFilters[B_Y*colorCache][B_Y * filtersPerThread]; // pre-load B_Y pixels from B_Y*filtersPerThread filters
    __shared__ float shImages[B_Y*colorCache][B_X * imgsPerThread]; // pre-load B_Y pixels from B_X*imgsPerThread images
    const int imgPixels = imgSize * imgSize;
    const int filterPixels = filterSize * filterSize;
    const int numFilterColors = numImgColors / numGroups;
    const int blocksPerModule = numFilters / (B_Y*filtersPerThread);
    const int moduleIdx = blockIdx.y / blocksPerModule;
    const int blockFilterIdx = filtersPerThread * B_Y * (blockIdx.y % blocksPerModule);
    const int numFiltersPerGroup = numFilters / numGroups;
    const int blockGroupIdx = blockFilterIdx / numFiltersPerGroup;

    const int numModules = numModulesX * numModulesX;
    const int blockColorIdx = numFilterColors * blockGroupIdx;

    const int tidx = threadIdx.y * B_X + threadIdx.x;

    const int imgLoadModPosY = (moduleIdx / numModulesX) * moduleStride;
    const int imgLoadModPosX = (moduleIdx % numModulesX) * moduleStride;

    const int shFilterLoadY = tidx / (B_Y * filtersPerThread);
    const int shFilterLoadX = tidx % (B_Y * filtersPerThread);
    const int myImgIdx = blockIdx.x * B_X * imgsPerThread + threadIdx.x;

    images += blockColorIdx * imgPixels * imgStride + myImgIdx;
    filters +=blockFilterIdx
            + shFilterLoadY * numFilters + shFilterLoadX;

    targets += moduleIdx * numImages
            + (blockFilterIdx + threadIdx.y) * numImages * numModules
            + myImgIdx;

    float prod[filtersPerThread][imgsPerThread];
    #pragma unroll
    for(int f = 0; f < filtersPerThread; f++) {
        #pragma unroll
        for(int g = 0; g < imgsPerThread; g++) {
            prod[f][g] = 0;
        }
    }

    for (int oc = 0; oc < numFilterColors; oc += colorCache) { // oc stands for outer color (loop)
        for (int p = 0; p < filterPixels; p += B_Y) {
            /*
             * Load B_Y pixels from B_Y*filtersPerThread filters
             */
            if (shFilterLoadY < B_Y) {
                #pragma unroll
                for (int p2 = 0; p2 < B_Y; p2 += B_X/filtersPerThread) {
                    if (p + p2 + shFilterLoadY < filterPixels) {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = filters[((oc+c) * filterPixels + p + p2) * numFilters];
                        }
                    } else {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shFilters[shFilterLoadY + p2 + c * B_Y][shFilterLoadX] = 0;
                        }
                    }
                }
            }

            /*
             * Load B_Y pixels from B_X*imgsPerThread images
             */
            int pixIdx = p + threadIdx.y;
            if (pixIdx < filterPixels) {
                const int x = paddingStart + imgLoadModPosX + pixIdx % filterSize;
                const int y = paddingStart + imgLoadModPosY + pixIdx / filterSize;
                if (y >= 0 && y < imgSize && x >= 0 && x < imgSize) {
                    #pragma unroll
                    for (int i = 0; i < imgsPerThread; i++) {
                        if (!checkImgBounds || myImgIdx + i * B_X < numImages) {
                            #pragma unroll
                            for (int c = 0; c < colorCache; c++) {
                                shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = images[imgStride * ((oc+c) * imgPixels + y * imgSize + x) + i * B_X];
                            }
                        } else {
                            #pragma unroll
                            for (int c = 0; c < colorCache; c++) {
                                shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                            }
                        }
                    }
                } else { // Padding
                    #pragma unroll
                    for (int i = 0; i < imgsPerThread; i++) {
                        #pragma unroll
                        for (int c = 0; c < colorCache; c++) {
                            shImages[threadIdx.y + c * B_Y][threadIdx.x + i * B_X] = 0;
                        }
                    }
                }
            }
            __syncthreads();
            #pragma unroll
            for (int i = 0; i < B_Y*colorCache; i++) {
                #pragma unroll
                for(int f = 0; f < filtersPerThread; f++) {
                    #pragma unroll
                    for(int g = 0; g < imgsPerThread; g++) {
                        prod[f][g] += shImages[i][g * B_X + threadIdx.x] * shFilters[i][threadIdx.y + f * B_Y];
                    }
                }

            }
            __syncthreads();
        }
    }

    if (scale) {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModules] = scaleTargets * targets[g * B_X + f * B_Y * numImages * numModules] + scaleOutputs * prod[f][g];
                }
            }
        }
    } else {
        #pragma unroll
        for (int g = 0; g < imgsPerThread; g++) {
            if (!checkImgBounds || myImgIdx + g * B_X < numImages) {
                #pragma unroll
                for (int f = 0; f < filtersPerThread; f++) {
                    targets[g * B_X + f * B_Y * numImages * numModules] = prod[f][g];
                }
            }
        }
    }
}

/*
 * images:      (numImgColors, imgPixels, numImages) with stride given
 * filters:     (numFilterColors, filterPixels, numFilters)
 *
 * targets:     (numFilters, numModules, numImages)
 *
 * images: The images matrix.
 * weights: The filters matrix.
 * targets: Result matrix.
 * numModulesX: number of filter applications in the x (or equivalently y) dimension. So the total
 *              number of modules will be the square of this number.
 * paddingStart: non-positive number indicating where the first filter should be applied.
 * moduleStride: stride between filter applications.
 * numColors: number of color channels in images and filters.
 * targetsOrder: how the output is to be laid out (see targets comment above)
 */
void convFilterActs(NVMatrix& images, NVMatrix& filters, NVMatrix& targets,
                          int numModulesX, int paddingStart, int moduleStride,
                          int numImgColors, int numGroups) {
    convFilterActs(images, filters, targets, numModulesX, paddingStart, moduleStride, numImgColors, numGroups, 0, 1);
}

void convFilterActs(NVMatrix& images, NVMatrix& filters, NVMatrix& targets,
                   int numModulesX, int paddingStart, int moduleStride,
                   int numImgColors, int numGroups,
                   float scaleTargets, float scaleOutput) {
    int numFilterColors = numImgColors / numGroups;      
    int numFilters = filters.getNumCols();
    int numModules = numModulesX * numModulesX;
    int numImages = images.getNumCols();
    int imgPixels = images.getNumRows()/numImgColors;
    int imgSize = int(sqrt(imgPixels));
    
    assert(numGroups > 1 || (numImgColors > 0 && (numImgColors <= 3 || numImgColors % 2 == 0)));
    assert(numGroups == 1 || numFilterColors % 2 == 0);
    assert(numFilters % (16 * numGroups) == 0);
    assert(numImgColors % numGroups == 0);
    assert(imgSize * imgSize == imgPixels);
    assert(images.getNumRows() == imgPixels * numImgColors);
    int numFiltersPerGroup = numFilters / numGroups;

    int imgStride = images.getStride(); // images does not need to be a contiguous matrix

    int filterPixels = filters.getNumRows() / numFilterColors;
    int filterSize = int(sqrt(filterPixels));
    assert(filterSize * filterSize == filterPixels);
    assert(filters.getNumRows() == numFilterColors* filterPixels);

    // These routines don't handle the case when only part of the image is visited in the convolution
    assert(paddingStart <= 0 && paddingStart + (numModules-1)*moduleStride + filterSize >= imgSize);
    assert(moduleStride <= filterSize);
    
    assert(!images.isTrans());
    assert(!filters.isTrans());
    assert(!targets.isTrans());

//    assert(numFiltersPerGroup % 16 == 0);

    assert(filters.isContiguous());
    assert(targets.isContiguous());

    dim3 blocks = numFiltersPerGroup % 32 == 0 ? dim3(DIVUP(numImages, 32 * 4), (numModules * numFilters) / (4 * 8))
                                               : dim3(DIVUP(numImages, 32 * 4), (numModules * numFilters) / (4 * 4));
    dim3 threads(32, 4);
    bool checkImgBounds = numImages % 128 != 0;
    if (scaleTargets == 0 && scaleOutput == 1) {
        targets.resize(numFilters * numModules, numImages);
    } else {
        assert(targets.getNumRows() == numFilters * numModules);
        assert(targets.getNumCols() == numImages);
    }
    
    if (numImgColors <= 3) {
        assert(numGroups == 1); // It has to be based on above definitions, but just to be sure.
        if (scaleTargets == 0 && scaleOutput == 1) { // don't scale
            if (numImgColors == 1) {
                if (checkImgBounds) {
                    if (numFilters % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 8, 1, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 4, 1, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    }
                } else {
                    if (numFilters % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 8, 1, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 4, 1, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    }
                }
            } else if (numImgColors == 2) {
                if (checkImgBounds) {
                    if (numFilters % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, false, true >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    }
                } else {
                    if (numFilters % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, false, false >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    }
                }
            }  else if (numImgColors == 3) {
                if (checkImgBounds) {
                     if (numFilters % 32 == 0) {
                         hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, false, true >, hipFuncCachePreferShared);
                         filterActs_YxX_color < 4, 32, 4, 8, 3, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                     numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                     } else {
                         hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, false, true >, hipFuncCachePreferShared);
                         filterActs_YxX_color < 4, 32, 4, 4, 3, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                     numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                     }
                } else {
                     if (numFilters % 32 == 0) {
                         hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, false, false >, hipFuncCachePreferShared);
                         filterActs_YxX_color < 4, 32, 4, 8, 3, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                     numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                     } else {
                         hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, false, false >, hipFuncCachePreferShared);
                         filterActs_YxX_color < 4, 32, 4, 4, 3, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                     numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                     }
                }
            }
        } else { // do scale
            if (numImgColors == 1) {
                if (checkImgBounds) {
                    if (numFilters % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, true, true >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 8, 1, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, true, true >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 4, 1, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    }
                } else {
                    if (numFilters % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 1, true, false >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 8, 1, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 1, true, false >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 4, 1, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    }
                }
            } else if (numImgColors == 2) {
                if (checkImgBounds) {
                    if (numFilters % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, true, true >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, true, true >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    }
                } else {
                    if (numFilters % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 2, true, false >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 2, true, false >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    }
                }
            }  else if (numImgColors == 3) {
                if (checkImgBounds) {
                    if (numFilters % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, true, true >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 8, 3, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, true, true >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 4, 3, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    }
                } else {
                    if (numFilters % 32 == 0) {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 8, 3, true, false >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 8, 3, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    } else {
                        hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_color< 4), 32, 4, 4, 3, true, false >, hipFuncCachePreferShared);
                        filterActs_YxX_color < 4, 32, 4, 4, 3, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                    numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, scaleTargets, scaleOutput);
                    }
                }
            }
        }
    } else {
        if (scaleTargets == 0 && scaleOutput == 1) { // don't scale
            if (checkImgBounds) {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse < 4, 32, 4, 8, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse < 4, 32, 4, 4, 2, false, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            } else {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse < 4, 32, 4, 8, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse < 4, 32, 4, 4, 2, false, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
        } else { // do scale
            if (checkImgBounds) {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse < 4, 32, 4, 8, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, true >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse < 4, 32, 4, 4, 2, true, true > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            } else {
                if (numFiltersPerGroup % 32 == 0) {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 8, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse < 4, 32, 4, 8, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                } else {
                    hipFuncSetCacheConfig(reinterpret_cast<const void*>(filterActs_YxX_sparse< 4), 32, 4, 4, 2, false, false >, hipFuncCachePreferShared);
                    filterActs_YxX_sparse < 4, 32, 4, 4, 2, true, false > <<<blocks, threads>>>(images.getDevData(), filters.getDevData(), targets.getDevData(),
                                numImages, numFilters, imgSize, filterSize, paddingStart, moduleStride, numModulesX, imgStride, numImgColors, numGroups, scaleTargets, scaleOutput);
                }
            }
        }
    }
    cutilCheckMsg("convFilterActs: kernel execution failed");
}