#include "hip/hip_runtime.h"
/* 
 * File:   layer.cu
 * Author: Alex Krizhevsky (akrizhevsky@gmail.com)
 *
 * Created on June 11, 2011, 6:18 AM
 */
#include <iostream>
#include "../include/layer_kernels.cuh"
#include "../include/layer.cuh"

using namespace std;

/* 
 * =======================
 * Layer
 * =======================
 */
/*
 * Static variables that controls whether the matrices storing the
 * unit activities and their gradients get destroyed after they are used.
 * 
 * Setting this to true might net a performance benefit of a few percent
 * while increasing memory consumption.
 */
bool Layer::_saveActs = true;
bool Layer::_saveActGrads = true;

Layer::Layer(PyObject* paramsDict, bool gradConsumer, bool gradProducer, bool trans) : 
             _gradConsumer(gradConsumer), _gradProducer(gradProducer), _trans(trans) {
    
    _name = PyString_AS_STRING(PyDict_GetItemString(paramsDict, "name"));
    _numGradProducersNext = 0;
}

void Layer::fpropNext() {
    for (int i = 0; i < _next.size(); i++) {
        _next[i]->fprop();
    }
}

void Layer::bpropPrev() {
    if (_gradProducer) {
        for (int i = 0; i < _prev.size(); i++) {
            if (_prev[i]->isGradConsumer()) {
                _prev[i]->bprop();
            }
        }
    }
}

void Layer::truncBwdActs() {
    if (!_saveActGrads) { 
        _actGrads.truncate();
    }
    if (!_saveActs) {
        _acts.truncate();
    }
}

void Layer::fprop() {
    _rcvdFInputs += 1;
    if (_rcvdFInputs == _prev.size()) {
        NVMatrixV v;
        for (int i = 0; i < _prev.size(); i++) {
            v.push_back(&_prev[i]->getActs());
        }
        fprop(v);
    }
}

void Layer::fprop(NVMatrix& v) {
    NVMatrixV vl;
    vl.push_back(&v);
    fprop(vl);
}

void Layer::fprop(NVMatrixV& v) {
    assert(v.size() == _prev.size());
    _rcvdFInputs = _prev.size();
    for (NVMatrixV::iterator it = v.begin(); it != v.end(); ++it) {
        (*it)->transpose(_trans);
    }
    _acts.transpose(_trans);
    _fprop(v);
    fpropNext();
}

void Layer::bprop() {
    _rcvdBInputs += 1;
    if (_rcvdBInputs == _numGradProducersNext) {
        bprop(_actGrads);
    }
}

void Layer::bprop(NVMatrix& v) {
    v.transpose(_trans);
    for (int i = 0; i < _prev.size(); i++) {
        _prev[i]->getActs().transpose(_trans);
        _prev[i]->getActGrads().transpose(_trans);
    }
    _acts.transpose(_trans);
    _bprop(v);
    truncBwdActs();
    bpropPrev();
}

void Layer::reset() {
    _rcvdFInputs = 0;
    _rcvdBInputs = 0;
}

const char* Layer::getName() {
    return _name;
}

int Layer::getRcvdFInputs() {
    return _rcvdFInputs;
}

int Layer::getRcvdBInputs() {
    return _rcvdBInputs;
}

void Layer::addNext(Layer* l) {
    _next.push_back(l);
    _numGradProducersNext += l->isGradProducer();
}

void Layer::addPrev(Layer* l) {
    _prev.push_back(l);
}

// Propagate gradient through this layer?
bool Layer::isGradConsumer() {
    return _gradConsumer;
}

// Does this layer produce gradient for layers below?
bool Layer::isGradProducer() {
    return _gradProducer;
}

vector<Layer*>& Layer::getPrev() {
    return _prev;
}

vector<Layer*>& Layer::getNext() {
    return _next;
}

NVMatrix& Layer::getActs() {
    return _acts;
}

NVMatrix& Layer::getActGrads() {
    return _actGrads;
}

void Layer::setCheckingGrads(bool v) {
    _checkingGrads = v;
}

/* 
 * =======================
 * FCLayer
 * =======================
 */

void FCLayer::multByInput(NVMatrix& input, int idx) {
    if (idx == 0) {
        input.rightMult(*_weights[idx], _acts);
    } else {
        _acts.addProduct(input, *_weights[idx]);
    }
}

FCLayer::FCLayer(PyObject* paramsDict) : Layer(paramsDict, true, true, true) {
    MatrixV* hWeights = getMatrixVec(PyDict_GetItemString(paramsDict, "weights"));
    MatrixV* hWeightsInc = getMatrixVec(PyDict_GetItemString(paramsDict, "weightsInc"));
    Matrix* hBiases = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "biases"));
    Matrix* hBiasesInc = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "biasesInc"));

    floatv* momW = getFloatVec(PyDict_GetItemString(paramsDict, "momW"));
    float momB = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "momB"));
    floatv* epsW = getFloatVec(PyDict_GetItemString(paramsDict, "epsW"));
    float epsB = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "epsB"));
    floatv* wc = getFloatVec(PyDict_GetItemString(paramsDict, "wc"));
    _weights.initialize(*hWeights, *hWeightsInc, *epsW, *wc, *momW, false);
    _biases.initialize(*hBiases, *hBiasesInc, epsB, 0, momB, true);

    char* neuronType = PyString_AS_STRING(PyDict_GetItemString(paramsDict, "neuron"));
    _neuron = &Neuron::makeNeuron(neuronType);
    assert(_biases.getNumRows() == 1);
}

void FCLayer::_fprop(NVMatrixV& v) {
    for (int i = 0; i < v.size(); i++) {
        multByInput(*v[i], i);
    }
    
    _acts.addVector(*_biases);
    _neuron->activate(_acts);
}

void FCLayer::_bprop(NVMatrix& v) {
    _neuron->computeInputGrads(v);
    v.sum(0, _biases.getGrads());
    for (int i = 0; i < _prev.size(); i++) {
        if (_prev[i]->isGradConsumer()) {
            NVMatrix& weights_T = _weights[i].getW().getTranspose();
            if (_prev[i]->getRcvdBInputs() == 0) {
                v.rightMult(weights_T, _prev[i]->getActGrads());
            } else {
                _prev[i]->getActGrads().addProduct(v, weights_T);
            }
            delete &weights_T;
        }
        NVMatrix& prevActs_T = _prev[i]->getActs().getTranspose();
        _weights[i].getInc().addProduct(prevActs_T, v,  (!_checkingGrads) * _weights[i].getMom(),
                                        _checkingGrads ? 1 : _weights[i].getEps() / v.getNumRows());
        delete &prevActs_T;
    }
}

void FCLayer::updateWeights(int numCases) {
    _weights.update(numCases);
    _biases.update(numCases);
}

void FCLayer::copyToCPU() {
    _weights.copyToCPU();
    _biases.copyToCPU();
}

void FCLayer::copyToGPU() {
    _weights.copyToGPU();
    _biases.copyToGPU();
}

void FCLayer::checkGradients(ConvNet* convNet) {
    for (int i = 0; i < _weights.getSize(); i++) {
        convNet->checkGradientsW(string(_name) + string(" weights[") + tostr(i) + string("]"), 0.1, _weights[i]);
    }
    convNet->checkGradientsW(string(_name) + string(" biases"), 0.01, _biases);
}

/* 
 * =======================
 * ConvLayer
 * =======================
 */
ConvLayer::ConvLayer(PyObject* paramsDict) : Layer(paramsDict, true, true, false) {
    Matrix* hWeights = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "weights"));
    Matrix* hWeightsInc = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "weightsInc"));
    Matrix* hBiases = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "biases"));
    Matrix* hBiasesInc = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "biasesInc"));
    
    float momW = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "momW"));
    float momB = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "momB"));
    float epsW = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "epsW"));
    float epsB = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "epsB"));
    float wc = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "wc"));
    
    _padding = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "padding"));
    _stride = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "stride"));
    _filterSize = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "filterSize"));
    _modulesX = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "modulesX"));
    _channels = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "channels"));
    _imgSize = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "imgSize"));
    _numFilters = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "numFilters"));
    _partialSum = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "partialSum"));
    _sharedBiases = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "sharedBiases"));

    _modules = _modulesX * _modulesX;
    _filterPixels = _filterSize * _filterSize;
    _imgPixels = _imgSize * _imgSize;
    
    _weights.initialize(*hWeights, *hWeightsInc, epsW, wc, momW, true);
    _biases.initialize(*hBiases, *hBiasesInc, epsB, 0, momB, true);

    char* neuronType = PyString_AS_STRING(PyDict_GetItemString(paramsDict, "neuron"));
    _neuron = &Neuron::makeNeuron(neuronType);
}

void ConvLayer::_fprop(NVMatrixV& v) {
    convFilterActs(*v[0], *_weights, _acts, _modulesX, _padding, _stride, _channels);
    if (_sharedBiases) {
        _acts.reshape(_numFilters, _acts.getNumElements() / _numFilters);
        _acts.addVector(*_biases);
        _acts.reshape(_numFilters * _modules, _acts.getNumElements() / (_numFilters * _modules));
    } else {
        _acts.addVector(*_biases);
    }
    
    _neuron->activate(_acts);
}

void ConvLayer::_bprop(NVMatrix& v) {
    _neuron->computeInputGrads(v);
    if (_sharedBiases) {
        v.reshape(_numFilters, v.getNumElements() / _numFilters);
        v.sum(1, _biases.getGrads());
        v.reshape(_numFilters * _modules, v.getNumElements() / (_numFilters * _modules));
    } else {
        v.sum(1, _biases.getGrads());
    }

    if (_prev[0]->isGradConsumer()) {
        float scaleTargets = _prev[0]->getRcvdBInputs() == 0 ? 0 : 1;
        convImgActs(v, *_weights, _prev[0]->getActGrads(), _imgSize, _padding, _stride, _channels, scaleTargets, 1);
    }
    if (_partialSum > 0 && _partialSum < _modules) {
        convWeightActs(_prev[0]->getActs(), v, _weightGradsTmp, _modulesX, _filterSize, _padding, _stride, _channels, 0, 1, _partialSum);
        _weightGradsTmp.reshape(_modules / _partialSum, _channels * _filterPixels * _numFilters);
        _weightGradsTmp.sum(0, _weights.getGrads());
        _weights.getGrads().reshape(_channels * _filterPixels, _numFilters);
    } else {
        convWeightActs(_prev[0]->getActs(), v, _weights.getGrads(), _modulesX, _filterSize, _padding, _stride, _channels);
    }
}

void ConvLayer::truncBwdActs() {
    Layer::truncBwdActs();
    if (!_saveActGrads) {
        _weightGradsTmp.truncate();
    }
}

void ConvLayer::updateWeights(int numCases) {
    _weights.update(numCases);
    _biases.update(numCases);
}

void ConvLayer::copyToCPU() {
    _weights.copyToCPU();
    _biases.copyToCPU();
}

void ConvLayer::copyToGPU() {
    _weights.copyToGPU();
    _biases.copyToGPU();
}

void ConvLayer::checkGradients(ConvNet* convNet) {
    convNet->checkGradientsW(string(_name) + string(" weights"), 0.01, _weights);
    convNet->checkGradientsW(string(_name) + string(" biases"), 0.02, _biases);
}

/* 
 * =======================
 * SoftmaxLayer
 * =======================
 */
SoftmaxLayer::SoftmaxLayer(PyObject* paramsDict) 
    : Layer(paramsDict, true, true, true) {
}

void SoftmaxLayer::_bprop(NVMatrix& v) {
    if (_prev[0]->isGradConsumer()) {
        assert(_prev.size() == 1);
        NVMatrix& target = _prev[0]->getActGrads();

        int numCases = _acts.getLeadingDim();
        int numOut = _acts.getFollowingDim();

        assert(v.getLeadingDim() == numCases && v.getFollowingDim() == numOut);

        dim3 threads(LOGREG_GRADS_THREADS_X, LOGREG_GRADS_THREADS_Y);
        dim3 blocks(DIVUP(numCases, LOGREG_GRADS_THREADS_X), DIVUP(numOut, LOGREG_GRADS_THREADS_Y));
        if (_prev[0]->getRcvdBInputs() == 0) {
            target.resize(_acts);
            kSoftmaxGrads<false><<<blocks, threads>>>(v.getDevData(), _acts.getDevData(), target.getDevData(), numCases, numOut);
        } else {
            kSoftmaxGrads<true><<<blocks, threads>>>(v.getDevData(), _acts.getDevData(), target.getDevData(), numCases, numOut);
        }

        cutilCheckMsg("kSoftmaxGrads: Kernel execution failed");
    }
}

void SoftmaxLayer::_fprop(NVMatrixV& v) {
    NVMatrix& input = *v[0];

    NVMatrix& max = input.max(1);
    input.addVector(max, -1, _acts);
    _acts.apply(NVMatrix::EXP);
    NVMatrix& sum = _acts.sum(1);
    _acts.eltwiseDivideByVector(sum);
    
    delete &max;
    delete &sum;
}

/* 
 * =======================
 * DataLayer
 * =======================
 */

DataLayer::DataLayer(PyObject* paramsDict) 
    : Layer(paramsDict, false, false, false) {
    _dataIdx = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "dataIdx"));
}

void DataLayer::fprop() {
    throw string("No dava given!");
}

void DataLayer::_fprop(NVMatrixV& data) {
    NVMatrix& d = *data[_dataIdx];
    // TODO: this is slightly inelegant because it creates a copy of the data structure
    // (though not of any GPU memory)
    _acts = d;
    // Make sure that _acts knows that it does not own its GPU memory
    _acts.setView(true);
}

void DataLayer::fprop(NVMatrixV& data) {
    _fprop(data);
    fpropNext();
}

void DataLayer::_bprop(NVMatrix& v) {
}

/* 
 * =====================
 * PoolLayer
 * =====================
 */
PoolLayer::PoolLayer(PyObject* paramsDict) 
    : Layer(paramsDict, true, true, false) {
    _channels = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "channels"));
    _sizeX = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "sizeX"));
    _start = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "start"));
    _stride = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "stride"));
    _outputsX = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "outputsX"));
    _imgSize = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "imgSize"));
    
    _pool = string(PyString_AS_STRING(PyDict_GetItemString(paramsDict, "pool")));
    if (_pool != string("max") && _pool != string("avg")) {
        throw string("Unknown pooling type ") + _pool;
    }
}

void PoolLayer::_fprop(NVMatrixV& v) {
    NVMatrix& images = *v[0];
    if (_pool == string("max")) {
        convLocalPool(images, _acts, _channels, _sizeX, _start, _stride, _outputsX, MaxPooler());
    } else if (_pool == string("avg")) {
        convLocalPool(images, _acts, _channels, _sizeX, _start, _stride, _outputsX, AvgPooler(_sizeX*_sizeX));
    }
}

void PoolLayer::_bprop(NVMatrix& v) {
    if (_prev[0]->isGradConsumer()) {
        float scaleTargets = _prev[0]->getRcvdBInputs() == 0 ? 0 : 1;
        if (_pool == string("max")) {
            convLocalMaxUndo(_prev[0]->getActs(), v, _acts, _prev[0]->getActGrads(), _sizeX, _start, _stride, _outputsX, scaleTargets, 1);
        } else if (_pool == string("avg")) {
            convLocalAvgUndo(v, _prev[0]->getActGrads(), _sizeX, _start, _stride, _outputsX, _imgSize, scaleTargets, 1);
        }
    }
}

/* 
 * =====================
 * ContrastNormLayer
 * =====================
 */
ContrastNormLayer::ContrastNormLayer(PyObject* paramsDict) 
    : Layer(paramsDict, true, true, false) {
    _channels = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "channels"));
    _sizeX = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "sizeX"));

    _scale = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "scale"));
}

void ContrastNormLayer::_fprop(NVMatrixV& v) {
    NVMatrix& images = *v[0];
    convContrastNorm(images, _denoms, _acts, _channels, _sizeX, _scale);
}

void ContrastNormLayer::_bprop(NVMatrix& v) {
    if (_prev[0]->isGradConsumer()) {
        float scaleTargets = _prev[0]->getRcvdBInputs() == 0 ? 0 : 1;
        convContrastNormUndo(v, _denoms, _prev[0]->getActs(), _acts, _prev[0]->getActGrads(), _channels, _sizeX, _scale, scaleTargets, 1);
    }
}

void ContrastNormLayer::truncBwdActs() {
    if (!_saveActGrads) { 
        _actGrads.truncate();
    }
    if (!_saveActs) {
        _acts.truncate();
        _denoms.truncate();
    }
}

/* 
 * =====================
 * CostLayer
 * =====================
 */
CostLayer::CostLayer(PyObject* paramsDict, bool gradConsumer, bool gradProducer, bool trans) 
    : Layer(paramsDict, gradConsumer, gradProducer, trans) {
    _coeff = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "coeff"));
    _gradProducer = _coeff != 0;
    _numGradProducersNext = 1;
}

double CostLayer::getCoeff() {
    return _coeff;
}

void CostLayer::bprop() {
    if (_coeff != 0) {
        Layer::bprop();
    }
}

doublev& CostLayer::getError() {
    doublev& v = *new doublev();
    v.insert(v.begin(), _err.begin(), _err.end());
    return v;
}

// TODO: make this a factory object
CostLayer& CostLayer::makeCostLayer(string& type, PyObject* paramsDict) {
    if (type == string("cost.logreg")) {
        return *new LogregCostLayer(paramsDict);
    }
    throw string("Unknown cost layer type ") + type;
}

/* 
 * =====================
 * LogregCostLayer
 * =====================
 */
LogregCostLayer::LogregCostLayer(PyObject* paramsDict) 
    : CostLayer(paramsDict, true, true, false) {
}

void LogregCostLayer::_fprop(NVMatrixV& v) {
    _err.clear();
    NVMatrix& labels = *v[0];
    NVMatrix& probs = *v[1];
    int numCases = labels.getNumElements();
    
    NVMatrix& trueLabelLogProbs = _acts, correctProbs;
    computeLogregCost(labels, probs, trueLabelLogProbs, correctProbs);
    _err.push_back(-trueLabelLogProbs.sum());
    _err.push_back(numCases - correctProbs.sum());
}

void LogregCostLayer::_bprop(NVMatrix& v) {
    NVMatrix& labels = _prev[0]->getActs();
    NVMatrix& probs = _prev[1]->getActs();
    NVMatrix& target = _prev[1]->getActGrads();

    computeLogregGrads(labels, probs, target, _prev[1]->getRcvdBInputs() > 0, _coeff);
}