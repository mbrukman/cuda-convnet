#include "hip/hip_runtime.h"
/* 
 * File:   layer.cu
 * Author: Alex Krizhevsky (akrizhevsky@gmail.com)
 *
 * Created on June 11, 2011, 6:18 AM
 */
#include <iostream>
#include "../include/layer_kernels.cuh"
#include "../include/layer.cuh"

using namespace std;

/* 
 * =======================
 * Layer
 * =======================
 */
bool Layer::saveBwdActs = false;

Layer::Layer(PyObject* paramsDict, ConvNet* convNet,
             bool gradConsumer, bool gradProducer, bool trans) : 
             _convNet(convNet), _gradConsumer(gradConsumer),
             _gradProducer(gradProducer), _trans(trans) {
    _name = PyString_AS_STRING((PyStringObject*)PyDict_GetItemString(paramsDict, "name"));
    // Connect backward links in graph for this layer

    intv* inputLayers = getIntVec((PyListObject*)PyDict_GetItemString(paramsDict, "inputs"));

    if (inputLayers != NULL) {
        for (int i = 0; i < inputLayers->size(); i++) {
            addPrev(&convNet->getLayer(inputLayers->at(i)));
        }
    }
    delete inputLayers;

    this->_numGradProducersNext = 0;
}

void Layer::fpropNext() {
    for (int i = 0; i < _next.size(); i++) {
        _next[i]->fprop();
    }
}

void Layer::truncActGrads() {
    if (!saveBwdActs) { 
        _actGrads.truncate();
    }
}

/*
 * Static method that controls whether the weight matrices storing the
 * unit activity gradients get destroyed after they are used.
 * 
 * Setting this to true might net a performance benefit of a few percent
 * while increasing memory consumption.
 */
void Layer::setSaveBwdActs(bool saveBwdActs) {
    Layer::saveBwdActs = saveBwdActs;
}

void Layer::fprop() {
    _rcvdFInputs += 1;
    if (_rcvdFInputs == _prev.size()) {
        NVMatrixV v;
        for (int i = 0; i < _prev.size(); i++) {
            v.push_back(&_prev[i]->getActs());
        }
        fprop(v);
    }
}

void Layer::fprop(NVMatrix& v) {
    NVMatrixV vl;
    vl.push_back(&v);
    fprop(vl);
}

void Layer::fprop(NVMatrixV& v) {
    assert(v.size() == _prev.size());
    _rcvdFInputs = _prev.size();
    for (NVMatrixV::iterator it = v.begin(); it != v.end(); ++it) {
        (*it)->transpose(_trans);
    }
    _acts.transpose(_trans);
    _fprop(v);
    fpropNext();
}

void Layer::bprop() {
    _rcvdBInputs += 1;
    if (_rcvdBInputs == _numGradProducersNext) {
        bprop(_actGrads);
    }
}

void Layer::bprop(NVMatrix& v) {
    v.transpose(_trans);
    for (int i = 0; i < _prev.size(); i++) {
        _prev[i]->getActs().transpose(_trans);
        _prev[i]->getActGrads().transpose(_trans);
    }
    _acts.transpose(_trans);
    _bprop(v);
}

void Layer::reset() {
    _rcvdFInputs = 0;
    _rcvdBInputs = 0;
}

const char* Layer::getName() {
    return _name;
}

int Layer::getRcvdFInputs() {
    return _rcvdFInputs;
}

int Layer::getRcvdBInputs() {
    return _rcvdBInputs;
}

void Layer::addNext(Layer* l) {
    _next.push_back(l);
    _numGradProducersNext += l->isGradProducer();
}

void Layer::addPrev(Layer* l) {
    _prev.push_back(l);
}

// Propagate gradient through this layer?
bool Layer::isGradConsumer() {
    return _gradConsumer;
}

// Does this layer produce gradient for layers below?
bool Layer::isGradProducer() {
    return _gradProducer;
}

vector<Layer*>& Layer::getPrev() {
    return _prev;
}

vector<Layer*>& Layer::getNext() {
    return _next;
}

NVMatrix& Layer::getActs() {
    return _acts;
}

NVMatrix& Layer::getActGrads() {
    return _actGrads;
}

/* 
 * =======================
 * FCLayer
 * =======================
 */

void FCLayer::multByInput(NVMatrix& input, int idx) {
    if (idx == 0) {
        input.rightMult(*_weights[idx], _acts);
    } else {
        _acts.addProduct(input, *_weights[idx]);
    }
}

FCLayer::FCLayer(PyObject* paramsDict, ConvNet* convNet) : Layer(paramsDict, convNet, true, true, true) {
    MatrixV* hWeights = getMatrixVec((PyListObject*)PyDict_GetItemString(paramsDict, "weights"));
    MatrixV* hWeightsInc = getMatrixVec((PyListObject*)PyDict_GetItemString(paramsDict, "weightsInc"));
    Matrix* hBiases = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "biases"));
    Matrix* hBiasesInc = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "biasesInc"));

    floatv* momW = getFloatVec((PyListObject*)PyDict_GetItemString(paramsDict, "momW"));
    float momB = PyFloat_AS_DOUBLE((PyFloatObject*)PyDict_GetItemString(paramsDict, "momB"));
    floatv* epsW = getFloatVec((PyListObject*)PyDict_GetItemString(paramsDict, "epsW"));
    float epsB = PyFloat_AS_DOUBLE((PyFloatObject*)PyDict_GetItemString(paramsDict, "epsB"));
    floatv* wc = getFloatVec((PyListObject*)PyDict_GetItemString(paramsDict, "wc"));
    _weights.initialize(*hWeights, *hWeightsInc, *epsW, *wc, *momW, false);
    _biases.initialize(*hBiases, *hBiasesInc, epsB, 0, momB, true);

    char* neuronType = PyString_AS_STRING((PyStringObject*)PyDict_GetItemString(paramsDict, "neuron"));
    _neuron = &Neuron::makeNeuron(neuronType);
    assert(_biases.getNumRows() == 1);
}

void FCLayer::_fprop(NVMatrixV& v) {
    for (int i = 0; i < v.size(); i++) {
        multByInput(*v[i], i);
    }
    
    _acts.addVector(*_biases);
    _neuron->activate(_acts);
}

void FCLayer::_bprop(NVMatrix& v) {
    _neuron->computeInputGrads(v);
    v.sum(0, _biases.getGrads());
    for (int i = 0; i < _prev.size(); i++) {
        if (_prev[i]->isGradConsumer()) {
            NVMatrix& weights_T = _weights[i].getW().getTranspose();
            if (_prev[i]->getRcvdBInputs() == 0) {
                v.rightMult(weights_T, _prev[i]->getActGrads());
            } else {
                _prev[i]->getActGrads().addProduct(v, weights_T);
            }
            delete &weights_T;
        }
        NVMatrix& prevActs_T = _prev[i]->getActs().getTranspose();
        _weights[i].getInc().addProduct(prevActs_T, v,  (!_convNet->isCheckingGrads()) * _weights[i].getMom(),
                                       _convNet->isCheckingGrads() ? 1 : _weights[i].getEps() / _convNet->getNumCases());
        delete &prevActs_T;
        
        _prev[i]->bprop();
    }
    truncActGrads();
}

void FCLayer::updateWeights() {
    _weights.update(_convNet->getNumCases());
    _biases.update(_convNet->getNumCases());
}

void FCLayer::copyToCPU() {
    _weights.copyToCPU();
    _biases.copyToCPU();
}

void FCLayer::copyToGPU() {
    _weights.copyToGPU();
    _biases.copyToGPU();
}

void FCLayer::checkGradients() {
    for (int i = 0; i < _weights.getSize(); i++) {
        _convNet->checkGradientsW(string(_name) + string(" weights[") + tostr(i) + string("]"), 0.1, _weights[i]);
    }
    _convNet->checkGradientsW(string(_name) + string(" biases"), 0.01, _biases);
}

/* 
 * =======================
 * ConvLayer
 * =======================
 */
ConvLayer::ConvLayer(PyObject* paramsDict, ConvNet* convNet) : Layer(paramsDict, convNet, true, true, false) {
    Matrix* hWeights = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "weights"));
    Matrix* hWeightsInc = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "weightsInc"));
    Matrix* hBiases = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "biases"));
    Matrix* hBiasesInc = new Matrix((PyArrayObject*)PyDict_GetItemString(paramsDict, "biasesInc"));
    
    float momW = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "momW"));
    float momB = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "momB"));
    float epsW = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "epsW"));
    float epsB = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "epsB"));
    float wc = PyFloat_AS_DOUBLE(PyDict_GetItemString(paramsDict, "wc"));
    
    _padding = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "padding"));
    _stride = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "stride"));
    _filterSize = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "filterSize"));
    _modulesX = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "modulesX"));
    _channels = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "channels"));
    _imgSize = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "imgSize"));
    _numFilters = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "numFilters"));
    _partialSum = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "partialSum"));
    _sharedBiases = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "sharedBiases"));

    _modules = _modulesX * _modulesX;
    _filterPixels = _filterSize * _filterSize;
    _imgPixels = _imgSize * _imgSize;
    
    _weights.initialize(*hWeights, *hWeightsInc, epsW, wc, momW, true);
    _biases.initialize(*hBiases, *hBiasesInc, epsB, 0, momB, true);

    char* neuronType = PyString_AS_STRING((PyStringObject*)PyDict_GetItemString(paramsDict, "neuron"));
    _neuron = &Neuron::makeNeuron(neuronType);
    assert(_prev.size() == 1); // Conv layer only has one input
}

void ConvLayer::_fprop(NVMatrixV& v) {
    convFilterActs(*v[0], *_weights, _acts, _modulesX, _padding, _stride, _channels, FILTER_MODULE_IMAGE);
    if (_sharedBiases) {
        _acts.reshape(_numFilters, _acts.getNumElements() / _numFilters);
        _acts.addVector(*_biases);
        _acts.reshape(_numFilters * _modules, _acts.getNumElements() / (_numFilters * _modules));
    } else {
        _acts.addVector(*_biases);
    }
    
    _neuron->activate(_acts);
}

void ConvLayer::_bprop(NVMatrix& v) {
    _neuron->computeInputGrads(v);
    if (_sharedBiases) {
        v.reshape(_numFilters, v.getNumElements() / _numFilters);
        v.sum(1, _biases.getGrads());
        v.reshape(_numFilters * _modules, v.getNumElements() / (_numFilters * _modules));
    } else {
        v.sum(1, _biases.getGrads());
    }

    if (_prev[0]->isGradConsumer()) {
        if (_prev[0]->getRcvdBInputs() == 0) {
            convImgActs(v, *_weights, _prev[0]->getActGrads(), _imgSize, _padding, _stride, _channels, FILTER_MODULE_IMAGE);
        } else {
            convImgActs(v, *_weights, _prev[0]->getActGrads(), _imgSize, _padding, _stride, _channels, 1, 1, FILTER_MODULE_IMAGE);
        }
    }
    if (_partialSum > 0 && _partialSum < _modules) {
        NVMatrix tmp;
        convWeightActs(_prev[0]->getActs(), v, tmp, _modulesX, _filterSize, _padding, _stride, _channels, 0, 1, FILTER_MODULE_IMAGE, _partialSum);
        tmp.reshape(_modules / _partialSum, _channels * _filterPixels * _numFilters);
        tmp.sum(0, _weights.getGrads());
        _weights.getGrads().reshape(_channels * _filterPixels, _numFilters);
    } else {
        convWeightActs(_prev[0]->getActs(), v, _weights.getGrads(), _modulesX, _filterSize, _padding, _stride, _channels, FILTER_MODULE_IMAGE);
    }
    
    truncActGrads();
    
    _prev[0]->bprop();
}

void ConvLayer::updateWeights() {
    _weights.update(_convNet->getNumCases());
    _biases.update(_convNet->getNumCases());
}

void ConvLayer::copyToCPU() {
    _weights.copyToCPU();
    _biases.copyToCPU();
}

void ConvLayer::copyToGPU() {
    _weights.copyToGPU();
    _biases.copyToGPU();
}

void ConvLayer::checkGradients() {
    _convNet->checkGradientsW(string(_name) + string(" weights"), 0.01, _weights);
    _convNet->checkGradientsW(string(_name) + string(" biases"), 0.02, _biases);
}

/* 
 * =======================
 * SoftmaxLayer
 * =======================
 */

SoftmaxLayer::SoftmaxLayer(PyObject* paramsDict, ConvNet* convNet) 
    : Layer(paramsDict, convNet, true, true, true) {
}

void SoftmaxLayer::_bprop(NVMatrix& v) {
    if (_prev[0]->isGradConsumer()) {
        
        assert(_prev.size() == 1);
        NVMatrix& target = _prev[0]->getActGrads();

        int numCases = _acts.getLeadingDim();
        int numOut = _acts.getFollowingDim();

        assert(v.getLeadingDim() == numCases && v.getFollowingDim() == numOut);

        dim3 threads(LOGREG_GRADS_THREADS_X, LOGREG_GRADS_THREADS_Y);
        dim3 blocks(DIVUP(numCases, LOGREG_GRADS_THREADS_X), DIVUP(numOut, LOGREG_GRADS_THREADS_Y));
        if (_prev[0]->getRcvdBInputs() == 0) {
            target.resize(_acts);
            kSoftmaxGrads<false><<<blocks, threads>>>(v.getDevData(), _acts.getDevData(), target.getDevData(), numCases, numOut);
        } else {
            kSoftmaxGrads<true><<<blocks, threads>>>(v.getDevData(), _acts.getDevData(), target.getDevData(), numCases, numOut);
        }

        cutilCheckMsg("kSoftmaxGrads: Kernel execution failed");

        truncActGrads();
        
        _prev[0]->bprop();
    }
}

void SoftmaxLayer::_fprop(NVMatrixV& v) {
    NVMatrix& input = *v[0];

    NVMatrix& max = input.max(1);
    input.addVector(max, -1, _acts);
    _acts.apply(NVMatrix::EXP);
    NVMatrix& sum = _acts.sum(1);
    _acts.eltwiseDivideByVector(sum);
    
    delete &max;
    delete &sum;
}

/* 
 * =======================
 * DataLayer
 * =======================
 */

DataLayer::DataLayer(PyObject* paramsDict, ConvNet* convNet) 
    : Layer(paramsDict, convNet, false, false, false) {
    _dataIdx = PyInt_AS_LONG((PyIntObject*)PyDict_GetItemString(paramsDict, "dataIdx"));
}

void DataLayer::fprop() {
    throw string("No dava given!");
}

void DataLayer::_fprop(NVMatrixV& data) {
    NVMatrix& d = *data[_dataIdx];
    // TODO: this is slightly inelegant because it creates a copy of the data structure
    // (though not of any GPU memory)
    _acts = d;
    // Make sure that _acts knows that it does not own its GPU memory
    _acts.setView(true);
}

void DataLayer::fprop(NVMatrixV& data) {
    _fprop(data);
    fpropNext();
}

void DataLayer::bprop() {

}

void DataLayer::_bprop(NVMatrix& v) {

}

/* 
 * =====================
 * PoolLayer
 * =====================
 */

PoolLayer::PoolLayer(PyObject* paramsDict, ConvNet* convNet) 
    : Layer(paramsDict, convNet, true, true, false) {
    _channels = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "channels"));
    _subsX = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "subsX"));
    _start = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "start"));
    _stride = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "stride"));
    _outputsX = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "outputsX"));
    _imgSize = PyInt_AS_LONG(PyDict_GetItemString(paramsDict, "imgSize"));
    
    _pool = string(PyString_AS_STRING((PyStringObject*)PyDict_GetItemString(paramsDict, "pool")));
    if (_pool != string("max") && _pool != string("avg")) {
        throw string("Unknown pooling type ") + _pool;
    }
}

void PoolLayer::_fprop(NVMatrixV& v) {
    NVMatrix& images = *v[0];
    if (_pool == string("max")) {
        convLocalPool(images, _acts, _channels, _subsX, _start, _stride, _outputsX, MaxAggregator());
    } else if (_pool == string("avg")) {
        convLocalPool(images, _acts, _channels, _subsX, _start, _stride, _outputsX, AvgAggregator(_subsX*_subsX));
    }
}

void PoolLayer::_bprop(NVMatrix& v) {
    if (_prev[0]->isGradConsumer()) {
        if (_pool == string("max")) {
            if (_prev[0]->getRcvdBInputs() == 0) {
                convLocalMaxUndo(_prev[0]->getActs(), v, _acts, _prev[0]->getActGrads(), _subsX, _start, _stride, _outputsX);
            } else {
                convLocalMaxUndo(_prev[0]->getActs(), v, _acts, _prev[0]->getActGrads(), _subsX, _start, _stride, _outputsX, 1, 1);
            }
        } else if (_pool == string("avg")) {
            if (_prev[0]->getRcvdBInputs() == 0) {
                convLocalAvgUndo(v, _prev[0]->getActGrads(), _subsX, _start, _stride, _outputsX, _imgSize);
            } else {
                convLocalAvgUndo(v, _prev[0]->getActGrads(), _subsX, _start, _stride, _outputsX, _imgSize, 1, 1);
            }
        }

        truncActGrads();
        _prev[0]->bprop();
    }
}

/* 
 * =====================
 * Cost
 * =====================
 */
Cost::Cost(PyObject* paramsDict, ConvNet* convNet, bool propagateGrad, bool gradProducer, bool trans) 
    : Layer(paramsDict, convNet, propagateGrad, gradProducer, trans) {
    _coeff = PyFloat_AS_DOUBLE((PyFloatObject*)PyDict_GetItemString(paramsDict, "coeff"));
    _gradProducer = _coeff != 0;
}

double Cost::getCoeff() {
    return _coeff;
}

void Cost::_bprop(NVMatrix& v) {
    throw string("Cost does not support _bprop(NVMatrix&)");
}

doublev& Cost::getError() {
    doublev* v = new doublev();
    for (doublev::const_iterator it = _err.begin(); it != _err.end(); ++it) {
        v->push_back(*it);
    }
    return *v;
}

/* 
 * =====================
 * LogregCost
 * =====================
 */

LogregCost::LogregCost(PyObject* paramsDict, ConvNet* convNet) 
    : Cost(paramsDict, convNet, true, true, false) {
}

void LogregCost::_fprop(NVMatrixV& v) {
    _err.clear();
    NVMatrix& labels = *v[0];
    NVMatrix& probs = *v[1];
    NVMatrix& maxProbs = probs.max(0);
    
    int caseStride = probs.getLeadingDim(); // num cases incl. padding
    int numOut = probs.getFollowingDim(); 
    NVMatrix trueLabelLogProbs(1, _convNet->getNumCases());
    NVMatrix correctProbs(1, _convNet->getNumCases());
    assert(labels.getNumElements() == caseStride);
    assert(labels.isContiguous());
    assert(probs.isContiguous());
    dim3 threads(LOGREG_ERR_THREADS_X, 1);
    dim3 blocks(DIVUP(_convNet->getNumCases(), LOGREG_ERR_THREADS_X), 1);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kLogregCost), hipFuncCachePreferL1);
    kLogregCost<<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), maxProbs.getDevData(),
                                     trueLabelLogProbs.getDevData(), correctProbs.getDevData(),
                                     _convNet->getNumCases(), caseStride, numOut);
    cutilCheckMsg("kLogregCost: Kernel execution failed");
    _err.push_back(-trueLabelLogProbs.sum());
    _err.push_back(_convNet->getNumCases() - correctProbs.sum());
    
    delete &maxProbs;
}

void LogregCost::bprop() {
    if (_coeff != 0) {
        NVMatrix& labels = _prev[0]->getActs();
        NVMatrix& probs = _prev[1]->getActs();
        NVMatrix& target = _prev[1]->getActGrads();
        int caseStride = probs.getLeadingDim(); // num cases incl. padding
        int numOut = probs.getFollowingDim();
        assert(labels.getNumElements() == caseStride);
        assert(probs.isContiguous());
        assert(target.isContiguous());
        assert(labels.isContiguous());
        dim3 threads(LOGREG_GRADS_THREADS_X, LOGREG_GRADS_THREADS_Y);
        dim3 blocks(DIVUP(caseStride, LOGREG_GRADS_THREADS_X), DIVUP(numOut, LOGREG_GRADS_THREADS_Y));
        if (_prev[1]->getRcvdBInputs() == 0) {
            target.resize(probs);
            kLogregCostGrads<false><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                         _convNet->getNumCases(), numOut, caseStride, _coeff);
        } else {
            kLogregCostGrads<true><<<blocks, threads>>>(probs.getDevData(), labels.getDevData(), target.getDevData(),
                                                        _convNet->getNumCases(), numOut, caseStride, _coeff);
        }
        cutilCheckMsg("kLogregCostGrads: Kernel execution failed");
        _prev[1]->bprop();
    }
}