#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <cutil_inline.h>
#include <iostream>

#include <layer_kernels.cuh>
#include <layer.cuh>
#include <data.cuh>
#include <util.cuh>
#include <cudaconv2.cuh>
#include <matrix.h>

using namespace std;

/* 
 * =======================
 * Layer
 * =======================
 */
/*
 * Static variables that control whether the matrices storing the
 * unit activities and their gradients get destroyed after they are used.
 * 
 * Setting these to true might net a performance benefit of a few percent
 * while increasing memory consumption.
 */
bool Layer::_saveActs = true;
bool Layer::_saveActsGrad = true;

Layer::Layer(PyObject* paramsDict, bool gradConsumer, bool gradProducer, bool trans) : 
             _gradConsumer(gradConsumer), _gradProducer(gradProducer), _trans(trans) {
    
    _name = pyDictGetString(paramsDict, "name");
    _type = pyDictGetString(paramsDict, "type");
    _numGradProducersNext = 0;
}

void Layer::fpropNext(PASS_TYPE passType) {
    for (int i = 0; i < _next.size(); i++) {
        _next[i]->fprop(passType);
    }
}

void Layer::truncBwdActs() {
    if (!_saveActsGrad) { 
        _actsGrad.truncate();
    }
    if (!_saveActs) {
        _outputs.truncate();
        getActs().truncate();
    }
}

void Layer::fprop(PASS_TYPE passType) {
    _rcvdFInputs += 1;
    if (_rcvdFInputs == _prev.size()) {
        NVMatrixV v;
        for (int i = 0; i < _prev.size(); i++) {
            v.push_back(&_prev[i]->getActs());
        }
        fprop(v, passType);
    }
}

void Layer::fprop(NVMatrix& v, PASS_TYPE passType) {
    NVMatrixV vl;
    vl.push_back(&v);
    fprop(vl, passType);
}

// TODO: make this remember v in a class variable, since it's necessary
// for gradient computation. At present bprop just assumes v == prev.getActs().
void Layer::fprop(NVMatrixV& v, PASS_TYPE passType) {
    assert(v.size() == _prev.size());
    _rcvdFInputs = _prev.size();
    for (NVMatrixV::iterator it = v.begin(); it != v.end(); ++it) {
        (*it)->transpose(_trans);
    }
    _outputs.transpose(_trans);
    getActs().transpose(_trans);
    fpropActs(v, passType);
    fpropNext(passType);
}

void Layer::bprop(PASS_TYPE passType) {
    if (_rcvdBInputs == _numGradProducersNext) {
        _rcvdBInputs++; // avoid doing bprop computation twice
        bprop(_actsGrad, passType);
    }
}

void Layer::bprop(NVMatrix& v, PASS_TYPE passType) {
    v.transpose(_trans);
    for (int i = 0; i < _prev.size(); i++) {
        _prev[i]->getActs().transpose(_trans);
        _prev[i]->getActsGrad().transpose(_trans);
    }
    getActs().transpose(_trans);
    
    bpropCommon(v, passType);
    
    if (_gradProducer) {
        for (int i = 0; i < _prev.size(); i++) {
            if (_prev[i]->isGradConsumer()) {
                bpropActs(v, i, _prev[i]->getRcvdBInputs() > 0 ? 1 : 0, passType);
                _prev[i]->incRcvdBInputs();
            }
        }
    }
    
    bpropWeights(v, passType);
    truncBwdActs();
    
    if (_gradProducer) {
        for (int i = 0; i < _prev.size(); i++) {
            if (_prev[i]->isGradConsumer()) {
                _prev[i]->bprop(passType);
            }
        }
    }
}

void Layer::reset() {
    _rcvdFInputs = 0;
    _rcvdBInputs = 0;
}

string& Layer::getName() {
    return _name;
}

string& Layer::getType() {
    return _type;
}

int Layer::getRcvdFInputs() {
    return _rcvdFInputs;
}

int Layer::getRcvdBInputs() {
    return _rcvdBInputs;
}

int Layer::incRcvdBInputs() {
    return ++_rcvdBInputs;
}

void Layer::addNext(Layer* l) {
    _next.push_back(l);
    _numGradProducersNext += l->isGradProducer();
}

void Layer::addPrev(Layer* l) {
    _prev.push_back(l);
}

// Propagate gradient through this layer?
bool Layer::isGradConsumer() {
    return _gradConsumer;
}

// Does this layer produce gradient for layers below?
bool Layer::isGradProducer() {
    return _gradProducer;
}

vector<Layer*>& Layer::getPrev() {
    return _prev;
}

vector<Layer*>& Layer::getNext() {
    return _next;
}

NVMatrix& Layer::getActs() {
    return _outputs;
}

NVMatrix& Layer::getActsGrad() {
    return _actsGrad;
}

/* 
 * =======================
 * WeightLayer
 * =======================
 */
WeightLayer::WeightLayer(PyObject* paramsDict, bool gradConsumer, bool gradProducer, bool trans) : 
    Layer(paramsDict, gradConsumer, gradProducer, trans) {
}

void WeightLayer::updateWeights(int numCases) {
    _allWeights.update(numCases);
}

void WeightLayer::copyToCPU() {
    _allWeights.copyToCPU();
}

void WeightLayer::copyToGPU() {
    _allWeights.copyToGPU();
}

/* 
 * =======================
 * FCLayer
 * =======================
 */
FCLayer::FCLayer(PyObject* paramsDict) : WeightLayer(paramsDict, true, true, true) {
    MatrixV* hWeights = pyDictGetMatrixV(paramsDict, "weights");
    MatrixV* hWeightsInc = pyDictGetMatrixV(paramsDict, "weightsInc");
    Matrix* hBiases = pyDictGetMatrix(paramsDict, "biases");
    Matrix* hBiasesInc = pyDictGetMatrix(paramsDict, "biasesInc");

    floatv* momW = pyDictGetFloatV(paramsDict, "momW");
    float momB = pyDictGetFloat(paramsDict, "momB");
    floatv* epsW = pyDictGetFloatV(paramsDict, "epsW");
    float epsB = pyDictGetFloat(paramsDict, "epsB");
    floatv* wc = pyDictGetFloatV(paramsDict, "wc");
    _weights.initialize(*hWeights, *hWeightsInc, *epsW, *wc, *momW, false);
    _biases.initialize(*hBiases, *hBiasesInc, epsB, 0, momB, true);

    _neuron = &Neuron::makeNeuron(PyDict_GetItemString(paramsDict, "neuron"), _outputs);
    
    _allWeights.addWeights(_weights);
    _allWeights.addWeights(_biases);
}

NVMatrix& FCLayer::getActs() {
    return _neuron->getActs();
}

void FCLayer::fpropActs(NVMatrixV& v, PASS_TYPE passType) {
    v[0]->rightMult(*_weights[0], _outputs);
    for (int i = 1; i < v.size(); i++) {
        _outputs.addProduct(*v[i], *_weights[i]);
    }
    _outputs.addVector(*_biases);
    _neuron->activate();
}

void FCLayer::bpropCommon(NVMatrix& v, PASS_TYPE passType) {
    _neuron->computeInputGrad(v);
}

void FCLayer::bpropActs(NVMatrix& v, int inpIdx, float scaleTargets, PASS_TYPE passType) {
    NVMatrix& weights_T = _weights[inpIdx].getW().getTranspose();
    if (scaleTargets == 0) {
        v.rightMult(weights_T, _prev[inpIdx]->getActsGrad());
    } else {
        _prev[inpIdx]->getActsGrad().addProduct(v, weights_T);
    }
    delete &weights_T;
}

void FCLayer::bpropWeights(NVMatrix& v, PASS_TYPE passType) {
    v.sum(0, _biases.getGrad());
    for (int i = 0; i < _prev.size(); i++) {
        NVMatrix& prevActs_T = _prev[i]->getActs().getTranspose();
        _weights[i].getInc().addProduct(prevActs_T, v,  (passType != PASS_GC) * _weights[i].getMom(),
                                        passType == PASS_GC ? 1 : _weights[i].getEps() / v.getNumRows());
        delete &prevActs_T;
    }
}

void FCLayer::checkGradients(ConvNet* convNet) {
    for (int i = 0; i < _weights.getSize(); i++) {
        convNet->checkGradient(_name + " weights[" + tostr(i) + "]", 0.1, _weights[i]);
    }
    convNet->checkGradient(_name + " biases", 0.01, _biases);
}

/* 
 * =======================
 * ConvLayer
 * =======================
 */
ConvLayer::ConvLayer(PyObject* paramsDict) : WeightLayer(paramsDict, true, true, false) {
    Matrix* hWeights = pyDictGetMatrix(paramsDict, "weights");
    Matrix* hWeightsInc = pyDictGetMatrix(paramsDict, "weightsInc");
    Matrix* hBiases = pyDictGetMatrix(paramsDict, "biases");
    Matrix* hBiasesInc = pyDictGetMatrix(paramsDict, "biasesInc");
    
    float momW = pyDictGetFloat(paramsDict, "momW");
    float momB = pyDictGetFloat(paramsDict, "momB");
    float epsW = pyDictGetFloat(paramsDict, "epsW");
    float epsB = pyDictGetFloat(paramsDict, "epsB");
    float wc = pyDictGetFloat(paramsDict, "wc");
    
    _padding = pyDictGetInt(paramsDict, "padding");
    _stride = pyDictGetInt(paramsDict, "stride");
    _filterSize = pyDictGetInt(paramsDict, "filterSize");
    _modulesX = pyDictGetInt(paramsDict, "modulesX");
    _channels = pyDictGetInt(paramsDict, "channels");
    _imgSize = pyDictGetInt(paramsDict, "imgSize");
    _numFilters = pyDictGetInt(paramsDict, "filters");
    _groups = pyDictGetInt(paramsDict, "groups");
    _partialSum = pyDictGetInt(paramsDict, "partialSum");
    _sharedBiases = pyDictGetInt(paramsDict, "sharedBiases");
    _randSparse = pyDictGetInt(paramsDict, "randSparse");
    _filterChannels = pyDictGetInt(paramsDict, "filterChannels");

    _modules = _modulesX * _modulesX;
    _filterPixels = _filterSize * _filterSize;
    _imgPixels = _imgSize * _imgSize;
    _overSample = (_groups * _filterChannels) / _channels;
    
    if (_randSparse) {
        _filterConns.hFilterConns = pyDictGetIntA(paramsDict, "filterConns");
    }
    
    _weights.initialize(*hWeights, *hWeightsInc, epsW, wc, momW, true);
    _biases.initialize(*hBiases, *hBiasesInc, epsB, 0, momB, true);

    _neuron = &Neuron::makeNeuron(PyDict_GetItemString(paramsDict, "neuron"), _outputs);
    
    _allWeights.addWeights(_weights);
    _allWeights.addWeights(_biases);
}

void ConvLayer::copyToGPU() {
    WeightLayer::copyToGPU();
    if (_randSparse) { // Copy vector that describes sparse random connectivity to GPU
        hipMalloc(&_filterConns.dFilterConns, sizeof(int) * _groups * _filterChannels);
        hipMemcpy(_filterConns.dFilterConns, _filterConns.hFilterConns, sizeof(int) * _groups * _filterChannels, hipMemcpyHostToDevice);
        cutilCheckMsg("hipMemcpy: failed");
    }
}

NVMatrix& ConvLayer::getActs() {
    return _neuron->getActs();
}

void ConvLayer::fpropActs(NVMatrixV& v, PASS_TYPE passType) {
    if (_randSparse) {
        convFilterActsSparse(*v[0], *_weights, _outputs, _filterConns.dFilterConns, _modulesX, _padding, _stride, _channels, _filterChannels, _groups);
    } else {
        convFilterActs(*v[0], *_weights, _outputs, _modulesX, _padding, _stride, _channels, _groups);
    }
    if (_sharedBiases) {
        _outputs.reshape(_numFilters, _outputs.getNumElements() / _numFilters);
        _outputs.addVector(*_biases);
        _outputs.reshape(_numFilters * _modules, _outputs.getNumElements() / (_numFilters * _modules));
    } else {
        _outputs.addVector(*_biases);
    }
    _neuron->activate();
}

void ConvLayer::bpropCommon(NVMatrix& v, PASS_TYPE passType) {
    _neuron->computeInputGrad(v);
}

void ConvLayer::bpropWeights(NVMatrix& v, PASS_TYPE passType) {
    if (_sharedBiases) {
        v.reshape(_numFilters, v.getNumElements() / _numFilters);
        v.sum(1, _biases.getGrad());
        v.reshape(_numFilters * _modules, v.getNumElements() / (_numFilters * _modules));
    } else {
        v.sum(1, _biases.getGrad());
    }
    
    NVMatrix& tgt = _partialSum > 0 ? _weightGradTmp : _weights.getGrad();
    if (_randSparse) {
        convWeightActsSparse(_prev[0]->getActs(), v, tgt, _filterConns.dFilterConns, _modulesX, _filterSize, _padding, _stride, _channels, _filterChannels, _groups, 0, 1, _partialSum);
    } else {
        convWeightActs(_prev[0]->getActs(), v, tgt, _modulesX, _filterSize, _padding, _stride, _channels, _groups, 0, 1, _partialSum);
    }
    if (_partialSum > 0) {
        _weightGradTmp.reshape(_modules / _partialSum, _filterChannels * _filterPixels * _numFilters);
        _weightGradTmp.sum(0, _weights.getGrad());
        _weights.getGrad().reshape(_filterChannels * _filterPixels, _numFilters);
    }
}

void ConvLayer::bpropActs(NVMatrix& v, int inpIdx, float scaleTargets, PASS_TYPE passType) {
    if (_randSparse) {
        if (_overSample > 1) {
            convImgActsSparse(v, *_weights, _actGradTmp, _filterConns.dFilterConns, _imgSize, _padding, _stride, _channels, _filterChannels, _groups, scaleTargets, 1);
            _actGradTmp.reshape(_overSample, _actGradTmp.getNumElements()/_overSample);
            _actGradTmp.sum(0, _prev[inpIdx]->getActsGrad());
            _prev[inpIdx]->getActsGrad().reshape(_prev[inpIdx]->getActsGrad().getNumElements()/_actGradTmp.getNumCols(), _actGradTmp.getNumCols());
        } else {
            convImgActsSparse(v, *_weights, _prev[inpIdx]->getActsGrad(), _filterConns.dFilterConns, _imgSize, _padding, _stride, _channels, _filterChannels, _groups, scaleTargets, 1);
        }
    } else {
        convImgActs(v, *_weights, _prev[inpIdx]->getActsGrad(), _imgSize, _padding, _stride, _channels, _groups, scaleTargets, 1);
    }
}

void ConvLayer::truncBwdActs() {
    Layer::truncBwdActs();
    if (!_saveActsGrad) {
        _weightGradTmp.truncate();
        _actGradTmp.truncate();
    }
}

void ConvLayer::checkGradients(ConvNet* convNet) {
    convNet->checkGradient(_name + " weights", 0.01, _weights);
    convNet->checkGradient(_name + " biases", 0.02, _biases);
}

/* 
 * =======================
 * SoftmaxLayer
 * =======================
 */
SoftmaxLayer::SoftmaxLayer(PyObject* paramsDict) : Layer(paramsDict, true, true, true) {
}

void SoftmaxLayer::bpropActs(NVMatrix& v, int inpIdx, float scaleTargets, PASS_TYPE passType) {
    bool doLogregGrad = _next.size() == 1 && _next[0]->getType() == "cost.logreg";
    if (doLogregGrad) {
        NVMatrix& labels = _next[0]->getPrev()[0]->getActs();
        float gradCoeff = dynamic_cast<CostLayer*>(_next[0])->getCoeff();
        computeLogregSoftmaxGrad(labels, _outputs, _prev[inpIdx]->getActsGrad(), scaleTargets == 1, gradCoeff);
    } else {
        computeSoftmaxGrad(_outputs, v, _prev[inpIdx]->getActsGrad(), scaleTargets == 1);
    }
}

void SoftmaxLayer::fpropActs(NVMatrixV& v, PASS_TYPE passType) {
    NVMatrix& input = *v[0];

    NVMatrix& max = input.max(1);
    input.addVector(max, -1, _outputs);
    _outputs.apply(NVMatrixOps::Exp());
    NVMatrix& sum = _outputs.sum(1);
    _outputs.eltwiseDivideByVector(sum);
    
    delete &max;
    delete &sum;
}

/* 
 * =======================
 * DataLayer
 * =======================
 */
DataLayer::DataLayer(PyObject* paramsDict) : Layer(paramsDict, false, false, false) {
    _dataIdx = pyDictGetInt(paramsDict, "dataIdx");
}

void DataLayer::fprop(PASS_TYPE passType) {
    throw string("No dava given!");
}

void DataLayer::fpropActs(NVMatrixV& data, PASS_TYPE passType) {
    NVMatrix& d = *data[_dataIdx];
    // TODO: this is slightly inelegant because it creates a copy of the data structure
    // (though not of any GPU memory)
    _outputs = d;
    // Make sure that _outputs knows that it does not own its GPU memory
    _outputs.setView(true);
}

void DataLayer::fprop(NVMatrixV& data, PASS_TYPE passType) {
    fpropActs(data, passType);
    fpropNext(passType);
}

/* 
 * =====================
 * PoolLayer
 * =====================
 */
PoolLayer::PoolLayer(PyObject* paramsDict, bool gradConsumer, bool gradProducer, bool trans) : Layer(paramsDict, gradConsumer, gradProducer, trans) {
    _channels = pyDictGetInt(paramsDict, "channels");
    _sizeX = pyDictGetInt(paramsDict, "sizeX");
    _start = pyDictGetInt(paramsDict, "start");
    _stride = pyDictGetInt(paramsDict, "stride");
    _outputsX = pyDictGetInt(paramsDict, "outputsX");
    _imgSize = pyDictGetInt(paramsDict, "imgSize");
    _pool = pyDictGetString(paramsDict, "pool");
}

PoolLayer& PoolLayer::makePoolLayer(PyObject* paramsDict) {
    string _pool = pyDictGetString(paramsDict, "pool");
    if (_pool == "max") {
        return *new MaxPoolLayer(paramsDict);
    } else if(_pool == "avg") {
        return *new AvgPoolLayer(paramsDict);
    }
    throw string("Unknown pooling layer type ") + _pool;
}

/* 
 * =====================
 * AvgPoolLayer
 * =====================
 */
AvgPoolLayer::AvgPoolLayer(PyObject* paramsDict) : PoolLayer(paramsDict, true, true, false) {
}

void AvgPoolLayer::fpropActs(NVMatrixV& v, PASS_TYPE passType) {
    convLocalPool(*v[0], _outputs, _channels, _sizeX, _start, _stride, _outputsX, AvgPooler(_sizeX*_sizeX));
}

void AvgPoolLayer::bpropActs(NVMatrix& v, int inpIdx, float scaleTargets, PASS_TYPE passType) {
    convLocalAvgUndo(v, _prev[inpIdx]->getActsGrad(), _sizeX, _start, _stride, _outputsX, _imgSize, scaleTargets, 1);
}

/* 
 * =====================
 * MaxPoolLayer
 * =====================
 */
MaxPoolLayer::MaxPoolLayer(PyObject* paramsDict) : PoolLayer(paramsDict, true, true, false) {
}

void MaxPoolLayer::fpropActs(NVMatrixV& v, PASS_TYPE passType) {
    convLocalPool(*v[0], _outputs, _channels, _sizeX, _start, _stride, _outputsX, MaxPooler());
}

void MaxPoolLayer::bpropActs(NVMatrix& v, int inpIdx, float scaleTargets, PASS_TYPE passType) {
    convLocalMaxUndo(_prev[inpIdx]->getActs(), v, _outputs, _prev[inpIdx]->getActsGrad(), _sizeX, _start, _stride, _outputsX, scaleTargets, 1);
}

/* 
 * =====================
 * ResponseNormLayer
 * =====================
 */
ResponseNormLayer::ResponseNormLayer(PyObject* paramsDict) : Layer(paramsDict, true, true, false) {
    _channels = pyDictGetInt(paramsDict, "channels");
    _sizeX = pyDictGetInt(paramsDict, "sizeX");

    _scale = pyDictGetFloat(paramsDict, "scale");
    _pow = pyDictGetFloat(paramsDict, "pow");
}

void ResponseNormLayer::fpropActs(NVMatrixV& v, PASS_TYPE passType) {
    convResponseNorm(*v[0], _denoms, _outputs, _channels, _sizeX, _scale, _pow);
}

void ResponseNormLayer::bpropActs(NVMatrix& v, int inpIdx, float scaleTargets, PASS_TYPE passType) {
    convResponseNormUndo(v, _denoms, _prev[inpIdx]->getActs(), _outputs, _prev[inpIdx]->getActsGrad(), _channels, _sizeX, _scale, _pow, scaleTargets, 1);
}

void ResponseNormLayer::truncBwdActs() {
    Layer::truncBwdActs();
    if (!_saveActs) {
        _denoms.truncate();
    }
}

/* 
 * =====================
 * ContrastNormLayer
 * =====================
 */
ContrastNormLayer::ContrastNormLayer(PyObject* paramsDict) : ResponseNormLayer(paramsDict) {
    _imgSize = pyDictGetInt(paramsDict, "imgSize");
}

void ContrastNormLayer::fpropActs(NVMatrixV& v, PASS_TYPE passType) {
    NVMatrix& images = *v[0];
    convLocalPool(images, _meanDiffs, _channels, _sizeX, -_sizeX/2, 1, _imgSize, AvgPooler(_sizeX*_sizeX));
    _meanDiffs.add(images, -1, 1);
    convContrastNorm(images, _meanDiffs, _denoms, _outputs, _channels, _sizeX, _scale, _pow);
}

void ContrastNormLayer::bpropActs(NVMatrix& v, int inpIdx, float scaleTargets, PASS_TYPE passType) {
    convContrastNormUndo(v, _denoms, _meanDiffs, _outputs, _prev[inpIdx]->getActsGrad(), _channels, _sizeX, _scale, _pow, scaleTargets, 1);
}

void ContrastNormLayer::truncBwdActs() {
    ResponseNormLayer::truncBwdActs();
    if (!_saveActs) {
        _meanDiffs.truncate();
    }
}

/* 
 * =====================
 * CostLayer
 * =====================
 */
CostLayer::CostLayer(PyObject* paramsDict, bool gradConsumer, bool gradProducer, bool trans) 
    : Layer(paramsDict, gradConsumer, gradProducer, trans) {
    _coeff = pyDictGetFloat(paramsDict, "coeff");
    _gradProducer = _coeff != 0;
}

float CostLayer::getCoeff() {
    return _coeff;
}

void CostLayer::bprop(PASS_TYPE passType) {
    if (_coeff != 0) {
        Layer::bprop(passType);
    }
}

doublev& CostLayer::getCost() {
    doublev& v = *new doublev();
    v.insert(v.begin(), _costv.begin(), _costv.end());
    return v;
}

CostLayer& CostLayer::makeCostLayer(string& type, PyObject* paramsDict) {
    if (type == "cost.logreg") {
        return *new LogregCostLayer(paramsDict);
    }
    throw string("Unknown cost layer type ") + type;
}

/* 
 * =====================
 * LogregCostLayer
 * =====================
 */
LogregCostLayer::LogregCostLayer(PyObject* paramsDict) : CostLayer(paramsDict, true, true, false) {
}

void LogregCostLayer::fpropActs(NVMatrixV& v, PASS_TYPE passType) {
    NVMatrix& labels = *v[0];
    NVMatrix& probs = *v[1];
    int numCases = labels.getNumElements();
    
    NVMatrix& trueLabelLogProbs = _outputs, correctProbs;
    computeLogregCost(labels, probs, trueLabelLogProbs, correctProbs);
    _costv.clear();
    _costv.push_back(-trueLabelLogProbs.sum());
    _costv.push_back(numCases - correctProbs.sum());
}

void LogregCostLayer::bpropActs(NVMatrix& v, int inpIdx, float scaleTargets, PASS_TYPE passType) {
    NVMatrix& labels = _prev[0]->getActs();
    NVMatrix& probs = _prev[inpIdx]->getActs();
    NVMatrix& target = _prev[inpIdx]->getActsGrad();
    // Numerical stability optimization: if the layer below me is a softmax layer, let it handle
    // the entire gradient computation to avoid multiplying and dividing by a near-zero quantity.
    bool doWork = _prev[inpIdx]->getNext().size() > 1 || _prev[inpIdx]->getType() != "softmax";
    if (doWork) {
        computeLogregGrad(labels, probs, target, scaleTargets == 1, _coeff);
    }
}