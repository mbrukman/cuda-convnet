#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2011, Alex Krizhevsky (akrizhevsky@gmail.com)
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification,
 * are permitted provided that the following conditions are met:
 *
 * - Redistributions of source code must retain the above copyright notice,
 *   this list of conditions and the following disclaimer.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice,
 *   this list of conditions and the following disclaimer in the documentation
 *   and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
 * NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE,
 * EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <iostream>
#include "../include/layer_kernels.cuh"
#include "../include/layer.cuh"

using namespace std;

/* 
 * =======================
 * Layer
 * =======================
 */
/*
 * Static variables that control whether the matrices storing the
 * unit activities and their gradients get destroyed after they are used.
 * 
 * Setting these to true might net a performance benefit of a few percent
 * while increasing memory consumption.
 */
bool Layer::_saveActs = true;
bool Layer::_saveActGrads = true;

/*
 * ConvNet sets this to true when gradient checking mode is enabled. Allows
 * the layers to change their computation when in that mode.
 */
bool Layer::_checkingGrads = false;

Layer::Layer(PyObject* paramsDict, bool gradConsumer, bool gradProducer, bool trans) : 
             _gradConsumer(gradConsumer), _gradProducer(gradProducer), _trans(trans) {
    
    _name = pyDictGetString(paramsDict, "name");
    _numGradProducersNext = 0;
}

void Layer::fpropNext() {
    for (int i = 0; i < _next.size(); i++) {
        _next[i]->fprop();
    }
}

void Layer::truncBwdActs() {
    if (!_saveActGrads) { 
        _actGrads.truncate();
    }
    if (!_saveActs) {
        _acts.truncate();
    }
}

void Layer::fprop() {
    _rcvdFInputs += 1;
    if (_rcvdFInputs == _prev.size()) {
        NVMatrixV v;
        for (int i = 0; i < _prev.size(); i++) {
            v.push_back(&_prev[i]->getActs());
        }
        fprop(v);
    }
}

void Layer::fprop(NVMatrix& v) {
    NVMatrixV vl;
    vl.push_back(&v);
    fprop(vl);
}

// TODO: make this remember v in a class variable, since it's necessary
// for gradient computation. At present bprop just assumes v == prev.getActs().
void Layer::fprop(NVMatrixV& v) {
    assert(v.size() == _prev.size());
    _rcvdFInputs = _prev.size();
    for (NVMatrixV::iterator it = v.begin(); it != v.end(); ++it) {
        (*it)->transpose(_trans);
    }
    _acts.transpose(_trans);
    fpropActs(v);
    fpropNext();
}

void Layer::bprop() {
    _rcvdBInputs += 1;
    if (_rcvdBInputs == _numGradProducersNext) {
        bprop(_actGrads);
    }
}

void Layer::bprop(NVMatrix& v) {
    v.transpose(_trans);
    for (int i = 0; i < _prev.size(); i++) {
        _prev[i]->getActs().transpose(_trans);
        _prev[i]->getActGrads().transpose(_trans);
    }
    _acts.transpose(_trans);
    
    bpropCommon(v);
    if (_gradProducer) {
        bpropActs(v);
    }
    bpropWeights(v);
    truncBwdActs();
    
    if (_gradProducer) {
        for (int i = 0; i < _prev.size(); i++) {
            if (_prev[i]->isGradConsumer()) {
                _prev[i]->bprop();
            }
        }
    }
}

void Layer::reset() {
    _rcvdFInputs = 0;
    _rcvdBInputs = 0;
}

string& Layer::getName() {
    return _name;
}

int Layer::getRcvdFInputs() {
    return _rcvdFInputs;
}

int Layer::getRcvdBInputs() {
    return _rcvdBInputs;
}

void Layer::addNext(Layer* l) {
    _next.push_back(l);
    _numGradProducersNext += l->isGradProducer();
}

void Layer::addPrev(Layer* l) {
    _prev.push_back(l);
}

// Propagate gradient through this layer?
bool Layer::isGradConsumer() {
    return _gradConsumer;
}

// Does this layer produce gradient for layers below?
bool Layer::isGradProducer() {
    return _gradProducer;
}

vector<Layer*>& Layer::getPrev() {
    return _prev;
}

vector<Layer*>& Layer::getNext() {
    return _next;
}

NVMatrix& Layer::getActs() {
    return _acts;
}

NVMatrix& Layer::getActGrads() {
    return _actGrads;
}

/* 
 * =======================
 * FCLayer
 * =======================
 */
FCLayer::FCLayer(PyObject* paramsDict) : Layer(paramsDict, true, true, true) {
    MatrixV* hWeights = pyDictGetMatrixV(paramsDict, "weights");
    MatrixV* hWeightsInc = pyDictGetMatrixV(paramsDict, "weightsInc");
    Matrix* hBiases = pyDictGetMatrix(paramsDict, "biases");
    Matrix* hBiasesInc = pyDictGetMatrix(paramsDict, "biasesInc");

    floatv* momW = pyDictGetFloatV(paramsDict, "momW");
    float momB = pyDictGetFloat(paramsDict, "momB");
    floatv* epsW = pyDictGetFloatV(paramsDict, "epsW");
    float epsB = pyDictGetFloat(paramsDict, "epsB");
    floatv* wc = pyDictGetFloatV(paramsDict, "wc");
    _weights.initialize(*hWeights, *hWeightsInc, *epsW, *wc, *momW, false);
    _biases.initialize(*hBiases, *hBiasesInc, epsB, 0, momB, true);

    string neuronType = pyDictGetString(paramsDict, "neuron");
    _neuron = &Neuron::makeNeuron(neuronType);
}

void FCLayer::fpropActs(NVMatrixV& v) {
    v[0]->rightMult(*_weights[0], _acts);
    for (int i = 1; i < v.size(); i++) {
        _acts.addProduct(*v[i], *_weights[i]);
    }
    
    _acts.addVector(*_biases);
    _neuron->activate(_acts);
}

void FCLayer::bpropCommon(NVMatrix& v) {
    _neuron->computeInputGrads(v);
}

void FCLayer::bpropActs(NVMatrix& v) {
    for (int i = 0; i < _prev.size(); i++) {
        if (_prev[i]->isGradConsumer()) {
            NVMatrix& weights_T = _weights[i].getW().getTranspose();
            if (_prev[i]->getRcvdBInputs() == 0) {
                v.rightMult(weights_T, _prev[i]->getActGrads());
            } else {
                _prev[i]->getActGrads().addProduct(v, weights_T);
            }
            delete &weights_T;
        }
    }
}

void FCLayer::bpropWeights(NVMatrix& v) {
    v.sum(0, _biases.getGrads());
    for (int i = 0; i < _prev.size(); i++) {
        NVMatrix& prevActs_T = _prev[i]->getActs().getTranspose();
        _weights[i].getInc().addProduct(prevActs_T, v,  (!_checkingGrads) * _weights[i].getMom(),
                                        _checkingGrads ? 1 : _weights[i].getEps() / v.getNumRows());
        delete &prevActs_T;
    }
}

void FCLayer::updateWeights(int numCases) {
    _weights.update(numCases);
    _biases.update(numCases);
}

void FCLayer::copyToCPU() {
    _weights.copyToCPU();
    _biases.copyToCPU();
}

void FCLayer::copyToGPU() {
    _weights.copyToGPU();
    _biases.copyToGPU();
}

void FCLayer::checkGradients(ConvNet* convNet) {
    for (int i = 0; i < _weights.getSize(); i++) {
        convNet->checkGradientsW(_name + " weights[" + tostr(i) + "]", 0.1, _weights[i]);
    }
    convNet->checkGradientsW(_name + " biases", 0.01, _biases);
}

/* 
 * =======================
 * ConvLayer
 * =======================
 */
ConvLayer::ConvLayer(PyObject* paramsDict) : Layer(paramsDict, true, true, false) {
    Matrix* hWeights = pyDictGetMatrix(paramsDict, "weights");
    Matrix* hWeightsInc = pyDictGetMatrix(paramsDict, "weightsInc");
    Matrix* hBiases = pyDictGetMatrix(paramsDict, "biases");
    Matrix* hBiasesInc = pyDictGetMatrix(paramsDict, "biasesInc");
    
    float momW = pyDictGetFloat(paramsDict, "momW");
    float momB = pyDictGetFloat(paramsDict, "momB");
    float epsW = pyDictGetFloat(paramsDict, "epsW");
    float epsB = pyDictGetFloat(paramsDict, "epsB");
    float wc = pyDictGetFloat(paramsDict, "wc");
    
    _padding = pyDictGetInt(paramsDict, "padding");
    _stride = pyDictGetInt(paramsDict, "stride");
    _filterSize = pyDictGetInt(paramsDict, "filterSize");
    _modulesX = pyDictGetInt(paramsDict, "modulesX");
    _channels = pyDictGetInt(paramsDict, "channels");
    _imgSize = pyDictGetInt(paramsDict, "imgSize");
    _numFilters = pyDictGetInt(paramsDict, "numFilters");
    _partialSum = pyDictGetInt(paramsDict, "partialSum");
    _sharedBiases = pyDictGetInt(paramsDict, "sharedBiases");

    _modules = _modulesX * _modulesX;
    _filterPixels = _filterSize * _filterSize;
    _imgPixels = _imgSize * _imgSize;
    
    _weights.initialize(*hWeights, *hWeightsInc, epsW, wc, momW, true);
    _biases.initialize(*hBiases, *hBiasesInc, epsB, 0, momB, true);

    string neuronType = pyDictGetString(paramsDict, "neuron");
    _neuron = &Neuron::makeNeuron(neuronType);
}

void ConvLayer::fpropActs(NVMatrixV& v) {
    convFilterActs(*v[0], *_weights, _acts, _modulesX, _padding, _stride, _channels);
    if (_sharedBiases) {
        _acts.reshape(_numFilters, _acts.getNumElements() / _numFilters);
        _acts.addVector(*_biases);
        _acts.reshape(_numFilters * _modules, _acts.getNumElements() / (_numFilters * _modules));
    } else {
        _acts.addVector(*_biases);
    }
    
    _neuron->activate(_acts);
}

void ConvLayer::bpropCommon(NVMatrix& v) {
    _neuron->computeInputGrads(v);
}

void ConvLayer::bpropWeights(NVMatrix& v) {
    if (_sharedBiases) {
        v.reshape(_numFilters, v.getNumElements() / _numFilters);
        v.sum(1, _biases.getGrads());
        v.reshape(_numFilters * _modules, v.getNumElements() / (_numFilters * _modules));
    } else {
        v.sum(1, _biases.getGrads());
    }
    if (_partialSum > 0 && _partialSum < _modules) {
        convWeightActs(_prev[0]->getActs(), v, _weightGradsTmp, _modulesX, _filterSize, _padding, _stride, _channels, 0, 1, _partialSum);
        _weightGradsTmp.reshape(_modules / _partialSum, _channels * _filterPixels * _numFilters);
        _weightGradsTmp.sum(0, _weights.getGrads());
        _weights.getGrads().reshape(_channels * _filterPixels, _numFilters);
    } else {
        convWeightActs(_prev[0]->getActs(), v, _weights.getGrads(), _modulesX, _filterSize, _padding, _stride, _channels);
    }
}

void ConvLayer::bpropActs(NVMatrix& v) {
    if (_prev[0]->isGradConsumer()) {
        float scaleTargets = _prev[0]->getRcvdBInputs() == 0 ? 0 : 1;
        convImgActs(v, *_weights, _prev[0]->getActGrads(), _imgSize, _padding, _stride, _channels, scaleTargets, 1);
    }
}

void ConvLayer::truncBwdActs() {
    Layer::truncBwdActs();
    if (!_saveActGrads) {
        _weightGradsTmp.truncate();
    }
}

void ConvLayer::updateWeights(int numCases) {
    _weights.update(numCases);
    _biases.update(numCases);
}

void ConvLayer::copyToCPU() {
    _weights.copyToCPU();
    _biases.copyToCPU();
}

void ConvLayer::copyToGPU() {
    _weights.copyToGPU();
    _biases.copyToGPU();
}

void ConvLayer::checkGradients(ConvNet* convNet) {
    convNet->checkGradientsW(_name + " weights", 0.01, _weights);
    convNet->checkGradientsW(_name + " biases", 0.02, _biases);
}

/* 
 * =======================
 * SoftmaxLayer
 * =======================
 */
SoftmaxLayer::SoftmaxLayer(PyObject* paramsDict) : Layer(paramsDict, true, true, true) {
}

void SoftmaxLayer::bpropActs(NVMatrix& v) {
    if (_prev[0]->isGradConsumer()) {
        computeSoftmaxGrads(_acts, v, _prev[0]->getActGrads(), _prev[0]->getRcvdBInputs() > 0);
    }
}

void SoftmaxLayer::fpropActs(NVMatrixV& v) {
    NVMatrix& input = *v[0];

    NVMatrix& max = input.max(1);
    input.addVector(max, -1, _acts);
    _acts.apply(NVMatrix::EXP);
    NVMatrix& sum = _acts.sum(1);
    _acts.eltwiseDivideByVector(sum);
    
    delete &max;
    delete &sum;
}

/* 
 * =======================
 * DataLayer
 * =======================
 */
DataLayer::DataLayer(PyObject* paramsDict) : Layer(paramsDict, false, false, false) {
    _dataIdx = pyDictGetInt(paramsDict, "dataIdx");
}

void DataLayer::fprop() {
    throw string("No dava given!");
}

void DataLayer::fpropActs(NVMatrixV& data) {
    NVMatrix& d = *data[_dataIdx];
    // TODO: this is slightly inelegant because it creates a copy of the data structure
    // (though not of any GPU memory)
    _acts = d;
    // Make sure that _acts knows that it does not own its GPU memory
    _acts.setView(true);
}

void DataLayer::fprop(NVMatrixV& data) {
    fpropActs(data);
    fpropNext();
}

/* 
 * =====================
 * PoolLayer
 * =====================
 */
PoolLayer::PoolLayer(PyObject* paramsDict) : Layer(paramsDict, true, true, false) {
    _channels = pyDictGetInt(paramsDict, "channels");
    _sizeX = pyDictGetInt(paramsDict, "sizeX");
    _start = pyDictGetInt(paramsDict, "start");
    _stride = pyDictGetInt(paramsDict, "stride");
    _outputsX = pyDictGetInt(paramsDict, "outputsX");
    _imgSize = pyDictGetInt(paramsDict, "imgSize");
    
    _pool = pyDictGetString(paramsDict, "pool");
    if (_pool != "max" && _pool != "avg") {
        throw string("Unknown pooling type ") + _pool;
    }
}

void PoolLayer::fpropActs(NVMatrixV& v) {
    NVMatrix& images = *v[0];
    if (_pool == "max") {
        convLocalPool(images, _acts, _channels, _sizeX, _start, _stride, _outputsX, MaxPooler());
    } else if (_pool == "avg") {
        convLocalPool(images, _acts, _channels, _sizeX, _start, _stride, _outputsX, AvgPooler(_sizeX*_sizeX));
    }
}

void PoolLayer::bpropActs(NVMatrix& v) {
    if (_prev[0]->isGradConsumer()) {
        float scaleTargets = _prev[0]->getRcvdBInputs() == 0 ? 0 : 1;
        if (_pool == "max") {
            convLocalMaxUndo(_prev[0]->getActs(), v, _acts, _prev[0]->getActGrads(), _sizeX, _start, _stride, _outputsX, scaleTargets, 1);
        } else if (_pool == "avg") {
            convLocalAvgUndo(v, _prev[0]->getActGrads(), _sizeX, _start, _stride, _outputsX, _imgSize, scaleTargets, 1);
        }
    }
}

/* 
 * =====================
 * ResponseNormLayer
 * =====================
 */
ResponseNormLayer::ResponseNormLayer(PyObject* paramsDict) : Layer(paramsDict, true, true, false) {
    _channels = pyDictGetInt(paramsDict, "channels");
    _sizeX = pyDictGetInt(paramsDict, "sizeX");

    _scale = pyDictGetFloat(paramsDict, "scale");
    _pow = pyDictGetFloat(paramsDict, "pow");
}

void ResponseNormLayer::fpropActs(NVMatrixV& v) {
    NVMatrix& images = *v[0];
    convResponseNorm(images, _denoms, _acts, _channels, _sizeX, _scale, _pow);
}

void ResponseNormLayer::bpropActs(NVMatrix& v) {
    if (_prev[0]->isGradConsumer()) {
        float scaleTargets = _prev[0]->getRcvdBInputs() == 0 ? 0 : 1;
        convResponseNormUndo(v, _denoms, _prev[0]->getActs(), _acts, _prev[0]->getActGrads(), _channels, _sizeX, _scale, _pow, scaleTargets, 1);
    }
}

void ResponseNormLayer::truncBwdActs() {
    Layer::truncBwdActs();
    if (!_saveActs) {
        _denoms.truncate();
    }
}

/* 
 * =====================
 * ContrastNormLayer
 * =====================
 */
ContrastNormLayer::ContrastNormLayer(PyObject* paramsDict) : ResponseNormLayer(paramsDict) {
    _imgSize = pyDictGetInt(paramsDict, "imgSize");
}

void ContrastNormLayer::fpropActs(NVMatrixV& v) {
    NVMatrix& images = *v[0];
    convLocalPool(images, _meanDiffs, _channels, _sizeX, -_sizeX/2, 1, _imgSize, AvgPooler(_sizeX*_sizeX));
    _meanDiffs.add(images, -1, 1);
    convContrastNorm(images, _meanDiffs, _denoms, _acts, _channels, _sizeX, _scale, _pow);
}

void ContrastNormLayer::bpropActs(NVMatrix& v) {
    if (_prev[0]->isGradConsumer()) {
        float scaleTargets = _prev[0]->getRcvdBInputs() == 0 ? 0 : 1;
        convContrastNormUndo(v, _denoms, _meanDiffs, _acts, _prev[0]->getActGrads(), _channels, _sizeX, _scale, _pow, scaleTargets, 1);
    }
}

void ContrastNormLayer::truncBwdActs() {
    ResponseNormLayer::truncBwdActs();
    if (!_saveActs) {
        _meanDiffs.truncate();
    }
}

/* 
 * =====================
 * CostLayer
 * =====================
 */
CostLayer::CostLayer(PyObject* paramsDict, bool gradConsumer, bool gradProducer, bool trans) 
    : Layer(paramsDict, gradConsumer, gradProducer, trans) {
    _coeff = pyDictGetFloat(paramsDict, "coeff");
    _gradProducer = _coeff != 0;
    _numGradProducersNext = 1;
}

double CostLayer::getCoeff() {
    return _coeff;
}

void CostLayer::bprop() {
    if (_coeff != 0) {
        Layer::bprop();
    }
}

doublev& CostLayer::getError() {
    doublev& v = *new doublev();
    v.insert(v.begin(), _err.begin(), _err.end());
    return v;
}

// TODO: make this a factory object
CostLayer& CostLayer::makeCostLayer(string& type, PyObject* paramsDict) {
    if (type == "cost.logreg") {
        return *new LogregCostLayer(paramsDict);
    }
    throw string("Unknown cost layer type ") + type;
}

/* 
 * =====================
 * LogregCostLayer
 * =====================
 */
LogregCostLayer::LogregCostLayer(PyObject* paramsDict) : CostLayer(paramsDict, true, true, false) {
}

void LogregCostLayer::fpropActs(NVMatrixV& v) {
    _err.clear();
    NVMatrix& labels = *v[0];
    NVMatrix& probs = *v[1];
    int numCases = labels.getNumElements();
    
    NVMatrix& trueLabelLogProbs = _acts, correctProbs;
    computeLogregCost(labels, probs, trueLabelLogProbs, correctProbs);
    _err.push_back(-trueLabelLogProbs.sum());
    _err.push_back(numCases - correctProbs.sum());
}

void LogregCostLayer::bpropActs(NVMatrix& v) {
    NVMatrix& labels = _prev[0]->getActs();
    NVMatrix& probs = _prev[1]->getActs();
    NVMatrix& target = _prev[1]->getActGrads();

    computeLogregGrads(labels, probs, target, _prev[1]->getRcvdBInputs() > 0, _coeff);
}